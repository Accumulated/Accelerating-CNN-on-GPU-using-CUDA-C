#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <hipsolver.h>
#include <hip/hip_runtime.h>


#include "/content/MBCONVS_float/Input_For_Stem_Layer.h"
#include "/content/MBCONVS_float/Stem/Stem_conv_parameters.h"
#include "/content/MBCONVS_float/functionsV2.h"
#include "/content/MBCONVS_float/CONFIG.h"
#include "/content/MBCONVS_float/Input_Matrix.h"
#include "/content/MBCONVS_float/KERNELSH.h"

#include "/content/MBCONVS_float/MBConv1_0/MBConv1_0_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MBConv1_0/MBConv1_0_project_conv_parameters.h"
#include "/content/MBCONVS_float/MBConv1_0/MBConv1_0_squeeze_excitation_parameters.h"

#include "/content/MBCONVS_float/MbConv6_1/MBConv6_1_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_1/MBConv6_1_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_1/MBConv6_1_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_1/MBConv6_1_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_2/MBConv6_2_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_2/MBConv6_2_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_2/MBConv6_2_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_2/MBConv6_2_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_3/MBConv6_3_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_3/MBConv6_3_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_3/MBConv6_3_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_3/MBConv6_3_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_4/MBConv6_4_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_4/MBConv6_4_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_4/MBConv6_4_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_4/MBConv6_4_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_5/MBConv6_5_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_5/MBConv6_5_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_5/MBConv6_5_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_5/MBConv6_5_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_6/MBConv6_6_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_6/MBConv6_6_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_6/MBConv6_6_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_6/MBConv6_6_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_7/MBConv6_7_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_7/MBConv6_7_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_7/MBConv6_7_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_7/MBConv6_7_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_8/MBConv6_8_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_8/MBConv6_8_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_8/MBConv6_8_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_8/MBConv6_8_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_9/MBConv6_9_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_9/MBConv6_9_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_9/MBConv6_9_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_9/MBConv6_9_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_10/MBConv6_10_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_10/MBConv6_10_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_10/MBConv6_10_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_10/MBConv6_10_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_11/MBConv6_11_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_11/MBConv6_11_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_11/MBConv6_11_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_11/MBConv6_11_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_12/MBConv6_12_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_12/MBConv6_12_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_12/MBConv6_12_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_12/MBConv6_12_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_13/MBConv6_13_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_13/MBConv6_13_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_13/MBConv6_13_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_13/MBConv6_13_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_14/MBConv6_14_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_14/MBConv6_14_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_14/MBConv6_14_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_14/MBConv6_14_project_conv_parameters.h"

#include "/content/MBCONVS_float/MbConv6_15/MBConv6_15_expansion_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_15/MBConv6_15_depthwise_conv_parameters.h"
#include "/content/MBCONVS_float/MbConv6_15/MBConv6_15_squeeze_excitation_parameters.h"
#include "/content/MBCONVS_float/MbConv6_15/MBConv6_15_project_conv_parameters.h"

#include "/content/MBCONVS_float/Head/Head_conv_parameters.h"


int MBCONV1_0_flag = 0;

int main()
{
  // 1. Define dimensions for input image.
  set_allocate_copy_array_Device(&DInput_Mat, Input_for_stem_conv,
                                 INPUT_IMAGE_HEIGHT, INPUT_IMAGE_WIDTH, 
                                 INPUT_IMAGE_DEPTH,
                                 "Input Image is allocated in device memory");  

  // 2. Get layers' filters ready
  set_allocate_copy_array_Device(&F_STEM, Stem_conv2d_weights,
                                 STEM_FILTER_HEIGHT, STEM_FILTER_WIDTH, 
                                 STEM_FILTER_DEPTH * STEM_FILTER_DENSITY,
                                 "Stem Filter  is allocated in device memory");
  
  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_1_0_EXPD_WEIGHTS, NULL, 
                            MBCONV_1_0_EXPD_F_HEIGHT,   MBCONV_1_0_EXPD_F_WIDTH, 
                            MBCONV_1_0_EXPD_F_DEPTH * MBCONV_1_0_EXPD_F_DENSITY,
                            &D_MBConv_1_0_DW_WEIGHTS, MBConv1_0_depthwise_conv_conv2d_weights, 
                            MBCONV_1_0_DW_F_HEIGHT, MBCONV_1_0_DW_F_WIDTH, 
                            MBCONV_1_0_DW_F_DEPTH * MBCONV_1_0_DW_F_DENSITY,
                            &D_MBConv_1_0_SQZ_1_WEIGHTS, MBConv1_0_squeeze_excitation1_conv2d_weights,
                            MBCONV_1_0_SQZ_1_F_HEIGHT, MBCONV_1_0_SQZ_1_F_WIDTH, 
                            MBCONV_1_0_SQZ_1_F_DEPTH * MBCONV_1_0_SQZ_1_F_DENSITY,
                            &D_MBConv_1_0_SQZ_2_WEIGHTS, MBConv1_0_squeeze_excitation2_conv2d_weights, 
                            MBCONV_1_0_SQZ_2_F_HEIGHT, MBCONV_1_0_SQZ_2_F_WIDTH, 
                            MBCONV_1_0_SQZ_2_F_DEPTH * MBCONV_1_0_SQZ_2_F_DENSITY,
                            &D_MBConv_1_0_PRJ_WEIGHTS, MBConv1_0_project_conv_conv2d_weights, 
                            MBCONV_1_0_PRJ_F_HEIGHT, MBCONV_1_0_PRJ_F_WIDTH, 
                            MBCONV_1_0_PRJ_F_DEPTH * MBCONV_1_0_PRJ_F_DENSITY); 

  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_1_EXPD_WEIGHTS, MBConv6_1_expansion_conv_conv2d_weights, 
                            MBCONV_6_1_EXPD_F_HEIGHT,   MBCONV_6_1_EXPD_F_WIDTH, 
                            MBCONV_6_1_EXPD_F_DEPTH * MBCONV_6_1_EXPD_F_DENSITY,
                            &D_MBConv_6_1_DW_WEIGHTS, MBConv6_1_depthwise_conv_conv2d_weights, 
                            MBCONV_6_1_DW_F_HEIGHT, MBCONV_6_1_DW_F_WIDTH, 
                            MBCONV_6_1_DW_F_DEPTH * MBCONV_6_1_DW_F_DENSITY,
                            &D_MBConv_6_1_SQZ_1_WEIGHTS, MBConv6_1_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_1_SQZ_1_F_HEIGHT, MBCONV_6_1_SQZ_1_F_WIDTH, 
                            MBCONV_6_1_SQZ_1_F_DEPTH * MBCONV_6_1_SQZ_1_F_DENSITY,
                            &D_MBConv_6_1_SQZ_2_WEIGHTS, MBConv6_1_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_1_SQZ_2_F_HEIGHT, MBCONV_6_1_SQZ_2_F_WIDTH, 
                            MBCONV_6_1_SQZ_2_F_DEPTH * MBCONV_6_1_SQZ_2_F_DENSITY,
                            &D_MBConv_6_1_PRJ_WEIGHTS, MBConv6_1_project_conv_conv2d_weights, 
                            MBCONV_6_1_PRJ_F_HEIGHT, MBCONV_6_1_PRJ_F_WIDTH, 
                            MBCONV_6_1_PRJ_F_DEPTH * MBCONV_6_1_PRJ_F_DENSITY); 

  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_2_EXPD_WEIGHTS, MBConv6_2_expansion_conv_conv2d_weights, 
                            MBCONV_6_2_EXPD_F_HEIGHT,   MBCONV_6_2_EXPD_F_WIDTH, 
                            MBCONV_6_2_EXPD_F_DEPTH * MBCONV_6_2_EXPD_F_DENSITY,
                            &D_MBConv_6_2_DW_WEIGHTS, MBConv6_2_depthwise_conv_conv2d_weights, 
                            MBCONV_6_2_DW_F_HEIGHT, MBCONV_6_2_DW_F_WIDTH, 
                            MBCONV_6_2_DW_F_DEPTH * MBCONV_6_2_DW_F_DENSITY,
                            &D_MBConv_6_2_SQZ_1_WEIGHTS, MBConv6_2_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_2_SQZ_1_F_HEIGHT, MBCONV_6_2_SQZ_1_F_WIDTH, 
                            MBCONV_6_2_SQZ_1_F_DEPTH * MBCONV_6_2_SQZ_1_F_DENSITY,
                            &D_MBConv_6_2_SQZ_2_WEIGHTS, MBConv6_2_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_2_SQZ_2_F_HEIGHT, MBCONV_6_2_SQZ_2_F_WIDTH, 
                            MBCONV_6_2_SQZ_2_F_DEPTH * MBCONV_6_2_SQZ_2_F_DENSITY,
                            &D_MBConv_6_2_PRJ_WEIGHTS, MBConv6_2_project_conv_conv2d_weights, 
                            MBCONV_6_2_PRJ_F_HEIGHT, MBCONV_6_2_PRJ_F_WIDTH, 
                            MBCONV_6_2_PRJ_F_DEPTH * MBCONV_6_2_PRJ_F_DENSITY);

  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_3_EXPD_WEIGHTS, MBConv6_3_expansion_conv_conv2d_weights, 
                            MBCONV_6_3_EXPD_F_HEIGHT,   MBCONV_6_3_EXPD_F_WIDTH, 
                            MBCONV_6_3_EXPD_F_DEPTH * MBCONV_6_3_EXPD_F_DENSITY,
                            &D_MBConv_6_3_DW_WEIGHTS, MBConv6_3_depthwise_conv_conv2d_weights, 
                            MBCONV_6_3_DW_F_HEIGHT, MBCONV_6_3_DW_F_WIDTH, 
                            MBCONV_6_3_DW_F_DEPTH * MBCONV_6_3_DW_F_DENSITY,
                            &D_MBConv_6_3_SQZ_1_WEIGHTS, MBConv6_3_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_3_SQZ_1_F_HEIGHT, MBCONV_6_3_SQZ_1_F_WIDTH, 
                            MBCONV_6_3_SQZ_1_F_DEPTH * MBCONV_6_3_SQZ_1_F_DENSITY,
                            &D_MBConv_6_3_SQZ_2_WEIGHTS, MBConv6_3_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_3_SQZ_2_F_HEIGHT, MBCONV_6_3_SQZ_2_F_WIDTH, 
                            MBCONV_6_3_SQZ_2_F_DEPTH * MBCONV_6_3_SQZ_2_F_DENSITY,
                            &D_MBConv_6_3_PRJ_WEIGHTS, MBConv6_3_project_conv_conv2d_weights, 
                            MBCONV_6_3_PRJ_F_HEIGHT, MBCONV_6_3_PRJ_F_WIDTH, 
                            MBCONV_6_3_PRJ_F_DEPTH * MBCONV_6_3_PRJ_F_DENSITY);


  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_4_EXPD_WEIGHTS, MBConv6_4_expansion_conv_conv2d_weights, 
                            MBCONV_6_4_EXPD_F_HEIGHT,   MBCONV_6_4_EXPD_F_WIDTH, 
                            MBCONV_6_4_EXPD_F_DEPTH * MBCONV_6_4_EXPD_F_DENSITY,
                            &D_MBConv_6_4_DW_WEIGHTS, MBConv6_4_depthwise_conv_conv2d_weights, 
                            MBCONV_6_4_DW_F_HEIGHT, MBCONV_6_4_DW_F_WIDTH, 
                            MBCONV_6_4_DW_F_DEPTH * MBCONV_6_4_DW_F_DENSITY,
                            &D_MBConv_6_4_SQZ_1_WEIGHTS, MBConv6_4_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_4_SQZ_1_F_HEIGHT, MBCONV_6_4_SQZ_1_F_WIDTH, 
                            MBCONV_6_4_SQZ_1_F_DEPTH * MBCONV_6_4_SQZ_1_F_DENSITY,
                            &D_MBConv_6_4_SQZ_2_WEIGHTS, MBConv6_4_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_4_SQZ_2_F_HEIGHT, MBCONV_6_4_SQZ_2_F_WIDTH, 
                            MBCONV_6_4_SQZ_2_F_DEPTH * MBCONV_6_4_SQZ_2_F_DENSITY,
                            &D_MBConv_6_4_PRJ_WEIGHTS, MBConv6_4_project_conv_conv2d_weights, 
                            MBCONV_6_4_PRJ_F_HEIGHT, MBCONV_6_4_PRJ_F_WIDTH, 
                            MBCONV_6_4_PRJ_F_DEPTH * MBCONV_6_4_PRJ_F_DENSITY);

  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_5_EXPD_WEIGHTS, MBConv6_5_expansion_conv_conv2d_weights, 
                            MBCONV_6_5_EXPD_F_HEIGHT,   MBCONV_6_5_EXPD_F_WIDTH, 
                            MBCONV_6_5_EXPD_F_DEPTH * MBCONV_6_5_EXPD_F_DENSITY,
                            &D_MBConv_6_5_DW_WEIGHTS, MBConv6_5_depthwise_conv_conv2d_weights, 
                            MBCONV_6_5_DW_F_HEIGHT, MBCONV_6_5_DW_F_WIDTH, 
                            MBCONV_6_5_DW_F_DEPTH * MBCONV_6_5_DW_F_DENSITY,
                            &D_MBConv_6_5_SQZ_1_WEIGHTS, MBConv6_5_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_5_SQZ_1_F_HEIGHT, MBCONV_6_5_SQZ_1_F_WIDTH, 
                            MBCONV_6_5_SQZ_1_F_DEPTH * MBCONV_6_5_SQZ_1_F_DENSITY,
                            &D_MBConv_6_5_SQZ_2_WEIGHTS, MBConv6_5_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_5_SQZ_2_F_HEIGHT, MBCONV_6_5_SQZ_2_F_WIDTH, 
                            MBCONV_6_5_SQZ_2_F_DEPTH * MBCONV_6_5_SQZ_2_F_DENSITY,
                            &D_MBConv_6_5_PRJ_WEIGHTS, MBConv6_5_project_conv_conv2d_weights, 
                            MBCONV_6_5_PRJ_F_HEIGHT, MBCONV_6_5_PRJ_F_WIDTH, 
                            MBCONV_6_5_PRJ_F_DEPTH * MBCONV_6_5_PRJ_F_DENSITY);
     
  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_6_EXPD_WEIGHTS, MBConv6_6_expansion_conv_conv2d_weights, 
                            MBCONV_6_6_EXPD_F_HEIGHT,   MBCONV_6_6_EXPD_F_WIDTH, 
                            MBCONV_6_6_EXPD_F_DEPTH * MBCONV_6_6_EXPD_F_DENSITY,
                            &D_MBConv_6_6_DW_WEIGHTS, MBConv6_6_depthwise_conv_conv2d_weights, 
                            MBCONV_6_6_DW_F_HEIGHT, MBCONV_6_6_DW_F_WIDTH, 
                            MBCONV_6_6_DW_F_DEPTH * MBCONV_6_6_DW_F_DENSITY,
                            &D_MBConv_6_6_SQZ_1_WEIGHTS, MBConv6_6_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_6_SQZ_1_F_HEIGHT, MBCONV_6_6_SQZ_1_F_WIDTH, 
                            MBCONV_6_6_SQZ_1_F_DEPTH * MBCONV_6_6_SQZ_1_F_DENSITY,
                            &D_MBConv_6_6_SQZ_2_WEIGHTS, MBConv6_6_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_6_SQZ_2_F_HEIGHT, MBCONV_6_6_SQZ_2_F_WIDTH, 
                            MBCONV_6_6_SQZ_2_F_DEPTH * MBCONV_6_6_SQZ_2_F_DENSITY,
                            &D_MBConv_6_6_PRJ_WEIGHTS, MBConv6_6_project_conv_conv2d_weights, 
                            MBCONV_6_6_PRJ_F_HEIGHT, MBCONV_6_6_PRJ_F_WIDTH, 
                            MBCONV_6_6_PRJ_F_DEPTH * MBCONV_6_6_PRJ_F_DENSITY);
     
  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_7_EXPD_WEIGHTS, MBConv6_7_expansion_conv_conv2d_weights, 
                            MBCONV_6_7_EXPD_F_HEIGHT,   MBCONV_6_7_EXPD_F_WIDTH, 
                            MBCONV_6_7_EXPD_F_DEPTH * MBCONV_6_7_EXPD_F_DENSITY,
                            &D_MBConv_6_7_DW_WEIGHTS, MBConv6_7_depthwise_conv_conv2d_weights, 
                            MBCONV_6_7_DW_F_HEIGHT, MBCONV_6_7_DW_F_WIDTH, 
                            MBCONV_6_7_DW_F_DEPTH * MBCONV_6_7_DW_F_DENSITY,
                            &D_MBConv_6_7_SQZ_1_WEIGHTS, MBConv6_7_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_7_SQZ_1_F_HEIGHT, MBCONV_6_7_SQZ_1_F_WIDTH, 
                            MBCONV_6_7_SQZ_1_F_DEPTH * MBCONV_6_7_SQZ_1_F_DENSITY,
                            &D_MBConv_6_7_SQZ_2_WEIGHTS, MBConv6_7_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_7_SQZ_2_F_HEIGHT, MBCONV_6_7_SQZ_2_F_WIDTH, 
                            MBCONV_6_7_SQZ_2_F_DEPTH * MBCONV_6_7_SQZ_2_F_DENSITY,
                            &D_MBConv_6_7_PRJ_WEIGHTS, MBConv6_7_project_conv_conv2d_weights, 
                            MBCONV_6_7_PRJ_F_HEIGHT, MBCONV_6_7_PRJ_F_WIDTH, 
                            MBCONV_6_7_PRJ_F_DEPTH * MBCONV_6_7_PRJ_F_DENSITY);


  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_8_EXPD_WEIGHTS, MBConv6_8_expansion_conv_conv2d_weights, 
                            MBCONV_6_8_EXPD_F_HEIGHT,   MBCONV_6_8_EXPD_F_WIDTH, 
                            MBCONV_6_8_EXPD_F_DEPTH * MBCONV_6_8_EXPD_F_DENSITY,
                            &D_MBConv_6_8_DW_WEIGHTS, MBConv6_8_depthwise_conv_conv2d_weights, 
                            MBCONV_6_8_DW_F_HEIGHT, MBCONV_6_8_DW_F_WIDTH, 
                            MBCONV_6_8_DW_F_DEPTH * MBCONV_6_8_DW_F_DENSITY,
                            &D_MBConv_6_8_SQZ_1_WEIGHTS, MBConv6_8_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_8_SQZ_1_F_HEIGHT, MBCONV_6_8_SQZ_1_F_WIDTH, 
                            MBCONV_6_8_SQZ_1_F_DEPTH * MBCONV_6_8_SQZ_1_F_DENSITY,
                            &D_MBConv_6_8_SQZ_2_WEIGHTS, MBConv6_8_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_8_SQZ_2_F_HEIGHT, MBCONV_6_8_SQZ_2_F_WIDTH, 
                            MBCONV_6_8_SQZ_2_F_DEPTH * MBCONV_6_8_SQZ_2_F_DENSITY,
                            &D_MBConv_6_8_PRJ_WEIGHTS, MBConv6_8_project_conv_conv2d_weights, 
                            MBCONV_6_8_PRJ_F_HEIGHT, MBCONV_6_8_PRJ_F_WIDTH, 
                            MBCONV_6_8_PRJ_F_DEPTH * MBCONV_6_8_PRJ_F_DENSITY);

  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_9_EXPD_WEIGHTS, MBConv6_9_expansion_conv_conv2d_weights, 
                            MBCONV_6_9_EXPD_F_HEIGHT,   MBCONV_6_9_EXPD_F_WIDTH, 
                            MBCONV_6_9_EXPD_F_DEPTH * MBCONV_6_9_EXPD_F_DENSITY,
                            &D_MBConv_6_9_DW_WEIGHTS, MBConv6_9_depthwise_conv_conv2d_weights, 
                            MBCONV_6_9_DW_F_HEIGHT, MBCONV_6_9_DW_F_WIDTH, 
                            MBCONV_6_9_DW_F_DEPTH * MBCONV_6_9_DW_F_DENSITY,
                            &D_MBConv_6_9_SQZ_1_WEIGHTS, MBConv6_9_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_9_SQZ_1_F_HEIGHT, MBCONV_6_9_SQZ_1_F_WIDTH, 
                            MBCONV_6_9_SQZ_1_F_DEPTH * MBCONV_6_9_SQZ_1_F_DENSITY,
                            &D_MBConv_6_9_SQZ_2_WEIGHTS, MBConv6_9_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_9_SQZ_2_F_HEIGHT, MBCONV_6_9_SQZ_2_F_WIDTH, 
                            MBCONV_6_9_SQZ_2_F_DEPTH * MBCONV_6_9_SQZ_2_F_DENSITY,
                            &D_MBConv_6_9_PRJ_WEIGHTS, MBConv6_9_project_conv_conv2d_weights, 
                            MBCONV_6_9_PRJ_F_HEIGHT, MBCONV_6_9_PRJ_F_WIDTH, 
                            MBCONV_6_9_PRJ_F_DEPTH * MBCONV_6_9_PRJ_F_DENSITY);


  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_10_EXPD_WEIGHTS, MBConv6_10_expansion_conv_conv2d_weights, 
                            MBCONV_6_10_EXPD_F_HEIGHT,   MBCONV_6_10_EXPD_F_WIDTH, 
                            MBCONV_6_10_EXPD_F_DEPTH * MBCONV_6_10_EXPD_F_DENSITY,
                            &D_MBConv_6_10_DW_WEIGHTS, MBConv6_10_depthwise_conv_conv2d_weights, 
                            MBCONV_6_10_DW_F_HEIGHT, MBCONV_6_10_DW_F_WIDTH, 
                            MBCONV_6_10_DW_F_DEPTH * MBCONV_6_10_DW_F_DENSITY,
                            &D_MBConv_6_10_SQZ_1_WEIGHTS, MBConv6_10_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_10_SQZ_1_F_HEIGHT, MBCONV_6_10_SQZ_1_F_WIDTH, 
                            MBCONV_6_10_SQZ_1_F_DEPTH * MBCONV_6_10_SQZ_1_F_DENSITY,
                            &D_MBConv_6_10_SQZ_2_WEIGHTS, MBConv6_10_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_10_SQZ_2_F_HEIGHT, MBCONV_6_10_SQZ_2_F_WIDTH, 
                            MBCONV_6_10_SQZ_2_F_DEPTH * MBCONV_6_10_SQZ_2_F_DENSITY,
                            &D_MBConv_6_10_PRJ_WEIGHTS, MBConv6_10_project_conv_conv2d_weights, 
                            MBCONV_6_10_PRJ_F_HEIGHT, MBCONV_6_10_PRJ_F_WIDTH, 
                            MBCONV_6_10_PRJ_F_DEPTH * MBCONV_6_10_PRJ_F_DENSITY);


  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_11_EXPD_WEIGHTS, MBConv6_11_expansion_conv_conv2d_weights, 
                            MBCONV_6_11_EXPD_F_HEIGHT,   MBCONV_6_11_EXPD_F_WIDTH, 
                            MBCONV_6_11_EXPD_F_DEPTH * MBCONV_6_11_EXPD_F_DENSITY,
                            &D_MBConv_6_11_DW_WEIGHTS, MBConv6_11_depthwise_conv_conv2d_weights, 
                            MBCONV_6_11_DW_F_HEIGHT, MBCONV_6_11_DW_F_WIDTH, 
                            MBCONV_6_11_DW_F_DEPTH * MBCONV_6_11_DW_F_DENSITY,
                            &D_MBConv_6_11_SQZ_1_WEIGHTS, MBConv6_11_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_11_SQZ_1_F_HEIGHT, MBCONV_6_11_SQZ_1_F_WIDTH, 
                            MBCONV_6_11_SQZ_1_F_DEPTH * MBCONV_6_11_SQZ_1_F_DENSITY,
                            &D_MBConv_6_11_SQZ_2_WEIGHTS, MBConv6_11_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_11_SQZ_2_F_HEIGHT, MBCONV_6_11_SQZ_2_F_WIDTH, 
                            MBCONV_6_11_SQZ_2_F_DEPTH * MBCONV_6_11_SQZ_2_F_DENSITY,
                            &D_MBConv_6_11_PRJ_WEIGHTS, MBConv6_11_project_conv_conv2d_weights, 
                            MBCONV_6_11_PRJ_F_HEIGHT, MBCONV_6_11_PRJ_F_WIDTH, 
                            MBCONV_6_11_PRJ_F_DEPTH * MBCONV_6_11_PRJ_F_DENSITY);

 
  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_12_EXPD_WEIGHTS, MBConv6_12_expansion_conv_conv2d_weights, 
                            MBCONV_6_12_EXPD_F_HEIGHT,   MBCONV_6_12_EXPD_F_WIDTH, 
                            MBCONV_6_12_EXPD_F_DEPTH * MBCONV_6_12_EXPD_F_DENSITY,
                            &D_MBConv_6_12_DW_WEIGHTS, MBConv6_12_depthwise_conv_conv2d_weights, 
                            MBCONV_6_12_DW_F_HEIGHT, MBCONV_6_12_DW_F_WIDTH, 
                            MBCONV_6_12_DW_F_DEPTH * MBCONV_6_12_DW_F_DENSITY,
                            &D_MBConv_6_12_SQZ_1_WEIGHTS, MBConv6_12_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_12_SQZ_1_F_HEIGHT, MBCONV_6_12_SQZ_1_F_WIDTH, 
                            MBCONV_6_12_SQZ_1_F_DEPTH * MBCONV_6_12_SQZ_1_F_DENSITY,
                            &D_MBConv_6_12_SQZ_2_WEIGHTS, MBConv6_12_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_12_SQZ_2_F_HEIGHT, MBCONV_6_12_SQZ_2_F_WIDTH, 
                            MBCONV_6_12_SQZ_2_F_DEPTH * MBCONV_6_12_SQZ_2_F_DENSITY,
                            &D_MBConv_6_12_PRJ_WEIGHTS, MBConv6_12_project_conv_conv2d_weights, 
                            MBCONV_6_12_PRJ_F_HEIGHT, MBCONV_6_12_PRJ_F_WIDTH, 
                            MBCONV_6_12_PRJ_F_DEPTH * MBCONV_6_12_PRJ_F_DENSITY);


  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_13_EXPD_WEIGHTS, MBConv6_13_expansion_conv_conv2d_weights, 
                            MBCONV_6_13_EXPD_F_HEIGHT,   MBCONV_6_13_EXPD_F_WIDTH, 
                            MBCONV_6_13_EXPD_F_DEPTH * MBCONV_6_13_EXPD_F_DENSITY,
                            &D_MBConv_6_13_DW_WEIGHTS, MBConv6_13_depthwise_conv_conv2d_weights, 
                            MBCONV_6_13_DW_F_HEIGHT, MBCONV_6_13_DW_F_WIDTH, 
                            MBCONV_6_13_DW_F_DEPTH * MBCONV_6_13_DW_F_DENSITY,
                            &D_MBConv_6_13_SQZ_1_WEIGHTS, MBConv6_13_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_13_SQZ_1_F_HEIGHT, MBCONV_6_13_SQZ_1_F_WIDTH, 
                            MBCONV_6_13_SQZ_1_F_DEPTH * MBCONV_6_13_SQZ_1_F_DENSITY,
                            &D_MBConv_6_13_SQZ_2_WEIGHTS, MBConv6_13_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_13_SQZ_2_F_HEIGHT, MBCONV_6_13_SQZ_2_F_WIDTH, 
                            MBCONV_6_13_SQZ_2_F_DEPTH * MBCONV_6_13_SQZ_2_F_DENSITY,
                            &D_MBConv_6_13_PRJ_WEIGHTS, MBConv6_13_project_conv_conv2d_weights, 
                            MBCONV_6_13_PRJ_F_HEIGHT, MBCONV_6_13_PRJ_F_WIDTH, 
                            MBCONV_6_13_PRJ_F_DEPTH * MBCONV_6_13_PRJ_F_DENSITY);


  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_14_EXPD_WEIGHTS, MBConv6_14_expansion_conv_conv2d_weights, 
                            MBCONV_6_14_EXPD_F_HEIGHT,   MBCONV_6_14_EXPD_F_WIDTH, 
                            MBCONV_6_14_EXPD_F_DEPTH * MBCONV_6_14_EXPD_F_DENSITY,
                            &D_MBConv_6_14_DW_WEIGHTS, MBConv6_14_depthwise_conv_conv2d_weights, 
                            MBCONV_6_14_DW_F_HEIGHT, MBCONV_6_14_DW_F_WIDTH, 
                            MBCONV_6_14_DW_F_DEPTH * MBCONV_6_14_DW_F_DENSITY,
                            &D_MBConv_6_14_SQZ_1_WEIGHTS, MBConv6_14_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_14_SQZ_1_F_HEIGHT, MBCONV_6_14_SQZ_1_F_WIDTH, 
                            MBCONV_6_14_SQZ_1_F_DEPTH * MBCONV_6_14_SQZ_1_F_DENSITY,
                            &D_MBConv_6_14_SQZ_2_WEIGHTS, MBConv6_14_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_14_SQZ_2_F_HEIGHT, MBCONV_6_14_SQZ_2_F_WIDTH, 
                            MBCONV_6_14_SQZ_2_F_DEPTH * MBCONV_6_14_SQZ_2_F_DENSITY,
                            &D_MBConv_6_14_PRJ_WEIGHTS, MBConv6_14_project_conv_conv2d_weights, 
                            MBCONV_6_14_PRJ_F_HEIGHT, MBCONV_6_14_PRJ_F_WIDTH, 
                            MBCONV_6_14_PRJ_F_DEPTH * MBCONV_6_14_PRJ_F_DENSITY);
     
  DEFINE_FILTERS_FOR_MBCONV(&D_MBConv_6_15_EXPD_WEIGHTS, MBConv6_15_expansion_conv_conv2d_weights, 
                            MBCONV_6_15_EXPD_F_HEIGHT,   MBCONV_6_15_EXPD_F_WIDTH, 
                            MBCONV_6_15_EXPD_F_DEPTH * MBCONV_6_15_EXPD_F_DENSITY,
                            &D_MBConv_6_15_DW_WEIGHTS, MBConv6_15_depthwise_conv_conv2d_weights, 
                            MBCONV_6_15_DW_F_HEIGHT, MBCONV_6_15_DW_F_WIDTH, 
                            MBCONV_6_15_DW_F_DEPTH * MBCONV_6_15_DW_F_DENSITY,
                            &D_MBConv_6_15_SQZ_1_WEIGHTS, MBConv6_15_squeeze_excitation1_conv2d_weights,
                            MBCONV_6_15_SQZ_1_F_HEIGHT, MBCONV_6_15_SQZ_1_F_WIDTH, 
                            MBCONV_6_15_SQZ_1_F_DEPTH * MBCONV_6_15_SQZ_1_F_DENSITY,
                            &D_MBConv_6_15_SQZ_2_WEIGHTS, MBConv6_15_squeeze_excitation2_conv2d_weights, 
                            MBCONV_6_15_SQZ_2_F_HEIGHT, MBCONV_6_15_SQZ_2_F_WIDTH, 
                            MBCONV_6_15_SQZ_2_F_DEPTH * MBCONV_6_15_SQZ_2_F_DENSITY,
                            &D_MBConv_6_15_PRJ_WEIGHTS, MBConv6_15_project_conv_conv2d_weights, 
                            MBCONV_6_15_PRJ_F_HEIGHT, MBCONV_6_15_PRJ_F_WIDTH, 
                            MBCONV_6_15_PRJ_F_DEPTH * MBCONV_6_15_PRJ_F_DENSITY);


  set_allocate_copy_array_Device(&HEAD_CONV_WEIGHTS, Head_conv2d_weights,
                                  HEAD_CONV_F_HEIGHT, HEAD_CONV_F_WIDTH, HEAD_CONV_F_DEPTH * HEAD_CONV_F_DENSITY,
                                "Head Filter  is allocated in device memory");   
 
  set_allocate_copy_array_Device(&HEAD_FC_WEIGHTS, Head_linear_weights,
                                HEAD_FC_F_HEIGHT, HEAD_FC_F_WIDTH, 1,
                                "Fully Connected weights matrix is allocated in device memory");  
  
  // Define bias matrices for all squeeze layers
  set_allocate_copy_array_Device(&MBConv6_15_SQZ_1_bias, MBConv6_15_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_15_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #15");  
  set_allocate_copy_array_Device(&MBConv6_14_SQZ_1_bias, MBConv6_14_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_14_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #14");
  set_allocate_copy_array_Device(&MBConv6_13_SQZ_1_bias, MBConv6_13_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_13_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #13");
  set_allocate_copy_array_Device(&MBConv6_12_SQZ_1_bias, MBConv6_12_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_12_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #12");
  set_allocate_copy_array_Device(&MBConv6_11_SQZ_1_bias, MBConv6_11_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_11_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #11");
  set_allocate_copy_array_Device(&MBConv6_10_SQZ_1_bias, MBConv6_10_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_10_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #10");  
  set_allocate_copy_array_Device(&MBConv6_9_SQZ_1_bias, MBConv6_9_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_9_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #9");
  set_allocate_copy_array_Device(&MBConv6_8_SQZ_1_bias, MBConv6_8_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_8_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #8");
  set_allocate_copy_array_Device(&MBConv6_7_SQZ_1_bias, MBConv6_7_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_7_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #7");
  set_allocate_copy_array_Device(&MBConv6_6_SQZ_1_bias, MBConv6_6_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_6_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #6");
  set_allocate_copy_array_Device(&MBConv6_5_SQZ_1_bias, MBConv6_5_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_5_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #5");  
  set_allocate_copy_array_Device(&MBConv6_4_SQZ_1_bias, MBConv6_4_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_4_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #4");
  set_allocate_copy_array_Device(&MBConv6_3_SQZ_1_bias, MBConv6_3_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_3_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #3");
  set_allocate_copy_array_Device(&MBConv6_2_SQZ_1_bias, MBConv6_2_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_2_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #2");
  set_allocate_copy_array_Device(&MBConv6_1_SQZ_1_bias, MBConv6_1_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv6_1_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #1");
  set_allocate_copy_array_Device(&MBConv1_0_SQZ_1_bias, MBConv1_0_squeeze_excitation1_conv2d_bias,
                                  sizeof(MBConv1_0_squeeze_excitation1_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 1 layer #0");   
  set_allocate_copy_array_Device(&MBConv6_15_SQZ_2_bias, MBConv6_15_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_15_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #15");  
  set_allocate_copy_array_Device(&MBConv6_14_SQZ_2_bias, MBConv6_14_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_14_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #14");
  set_allocate_copy_array_Device(&MBConv6_13_SQZ_2_bias, MBConv6_13_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_13_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #13");
  set_allocate_copy_array_Device(&MBConv6_12_SQZ_2_bias, MBConv6_12_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_12_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #12");
  set_allocate_copy_array_Device(&MBConv6_11_SQZ_2_bias, MBConv6_11_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_11_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #11");
  set_allocate_copy_array_Device(&MBConv6_10_SQZ_2_bias, MBConv6_10_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_10_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #10");  
  set_allocate_copy_array_Device(&MBConv6_9_SQZ_2_bias, MBConv6_9_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_9_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #9");
  set_allocate_copy_array_Device(&MBConv6_8_SQZ_2_bias, MBConv6_8_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_8_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #8");
  set_allocate_copy_array_Device(&MBConv6_7_SQZ_2_bias, MBConv6_7_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_7_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #7");
  set_allocate_copy_array_Device(&MBConv6_6_SQZ_2_bias, MBConv6_6_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_6_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #6");
  set_allocate_copy_array_Device(&MBConv6_5_SQZ_2_bias, MBConv6_5_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_5_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #5");  
  set_allocate_copy_array_Device(&MBConv6_4_SQZ_2_bias, MBConv6_4_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_4_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #4");
  set_allocate_copy_array_Device(&MBConv6_3_SQZ_2_bias, MBConv6_3_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_3_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #3");
  set_allocate_copy_array_Device(&MBConv6_2_SQZ_2_bias, MBConv6_2_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_2_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #2");
  set_allocate_copy_array_Device(&MBConv6_1_SQZ_2_bias, MBConv6_1_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv6_1_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #1");
  set_allocate_copy_array_Device(&MBConv1_0_SQZ_2_bias, MBConv1_0_squeeze_excitation2_conv2d_bias,
                                  sizeof(MBConv1_0_squeeze_excitation2_conv2d_bias)/sizeof(float), 1, 1,
                                  "Bias for squeeze 2 layer #0");    

// 3. Define BN mean,variance, weights and bias
MBCONV1_0_flag = 1;

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv1_0_EXPD_BN_MEAN,      NULL, 0,
  &MBConv1_0_EXPD_BN_VARIANCE,	NULL, 0,
  &MBConv1_0_EXPD_BN_WEIGHTS,		NULL, 0,
  &MBConv1_0_EXPD_BN_BIAS,			NULL, 0,

  &MBConv1_0_DW_BN_MEAN,        MBConv1_0_depthwise_conv_BN_mean,		  sizeof(MBConv1_0_depthwise_conv_BN_mean) / sizeof(float), 		
  &MBConv1_0_DW_BN_VARIANCE,		MBConv1_0_depthwise_conv_BN_variance,	sizeof(MBConv1_0_depthwise_conv_BN_variance) / sizeof(float),
  &MBConv1_0_DW_BN_WEIGHTS,     MBConv1_0_depthwise_conv_BN_weights,	sizeof(MBConv1_0_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv1_0_DW_BN_BIAS,				MBConv1_0_depthwise_conv_BN_bias,		  sizeof(MBConv1_0_depthwise_conv_BN_bias) / sizeof(float),

  &MBConv1_0_PRJ_BN_MEAN,       MBConv1_0_project_conv_BN_mean,			  sizeof(MBConv1_0_project_conv_BN_mean) / sizeof(float),
  &MBConv1_0_PRJ_BN_VARIANCE,		MBConv1_0_project_conv_BN_variance,		sizeof(MBConv1_0_project_conv_BN_variance) / sizeof(float),
  &MBConv1_0_PRJ_BN_WEIGHTS,    MBConv1_0_project_conv_BN_weights,		sizeof(MBConv1_0_project_conv_BN_weights) / sizeof(float),
  &MBConv1_0_PRJ_BN_BIAS,				MBConv1_0_project_conv_BN_bias, 		  sizeof(MBConv1_0_project_conv_BN_bias) / sizeof(float));

MBCONV1_0_flag = 0;

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_1_EXPD_BN_MEAN,      MBConv6_1_expansion_conv_BN_mean,		  sizeof(MBConv6_1_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_1_EXPD_BN_VARIANCE,	MBConv6_1_expansion_conv_BN_variance,	sizeof(MBConv6_1_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_1_EXPD_BN_WEIGHTS,   MBConv6_1_expansion_conv_BN_weights,	sizeof(MBConv6_1_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_1_EXPD_BN_BIAS,			MBConv6_1_expansion_conv_BN_bias,		  sizeof(MBConv6_1_expansion_conv_BN_bias) / sizeof(float),

  &MBConv6_1_DW_BN_MEAN,        MBConv6_1_depthwise_conv_BN_mean,		  sizeof(MBConv6_1_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_1_DW_BN_VARIANCE,		MBConv6_1_depthwise_conv_BN_variance,	sizeof(MBConv6_1_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_1_DW_BN_WEIGHTS,     MBConv6_1_depthwise_conv_BN_weights,	sizeof(MBConv6_1_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_1_DW_BN_BIAS,				MBConv6_1_depthwise_conv_BN_bias,		  sizeof(MBConv6_1_depthwise_conv_BN_bias) / sizeof(float),

  &MBConv6_1_PRJ_BN_MEAN,       MBConv6_1_project_conv_BN_mean,			  sizeof(MBConv6_1_project_conv_BN_mean) / sizeof(float),
  &MBConv6_1_PRJ_BN_VARIANCE,		MBConv6_1_project_conv_BN_variance,		sizeof(MBConv6_1_project_conv_BN_variance) / sizeof(float),
  &MBConv6_1_PRJ_BN_WEIGHTS,    MBConv6_1_project_conv_BN_weights,		sizeof(MBConv6_1_project_conv_BN_weights) / sizeof(float),
  &MBConv6_1_PRJ_BN_BIAS,				MBConv6_1_project_conv_BN_bias, 		  sizeof(MBConv6_1_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_2_EXPD_BN_MEAN,      MBConv6_2_expansion_conv_BN_mean,		  sizeof(MBConv6_2_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_2_EXPD_BN_VARIANCE,	MBConv6_2_expansion_conv_BN_variance,	sizeof(MBConv6_2_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_2_EXPD_BN_WEIGHTS,   MBConv6_2_expansion_conv_BN_weights,	sizeof(MBConv6_2_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_2_EXPD_BN_BIAS,			MBConv6_2_expansion_conv_BN_bias,		  sizeof(MBConv6_2_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_2_DW_BN_MEAN,        MBConv6_2_depthwise_conv_BN_mean,		  sizeof(MBConv6_2_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_2_DW_BN_VARIANCE,		MBConv6_2_depthwise_conv_BN_variance,	sizeof(MBConv6_2_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_2_DW_BN_WEIGHTS,     MBConv6_2_depthwise_conv_BN_weights,	sizeof(MBConv6_2_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_2_DW_BN_BIAS,			  MBConv6_2_depthwise_conv_BN_bias,		  sizeof(MBConv6_2_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_2_PRJ_BN_MEAN,       MBConv6_2_project_conv_BN_mean,			  sizeof(MBConv6_2_project_conv_BN_mean) / sizeof(float),
  &MBConv6_2_PRJ_BN_VARIANCE,		MBConv6_2_project_conv_BN_variance,		sizeof(MBConv6_2_project_conv_BN_variance) / sizeof(float),
  &MBConv6_2_PRJ_BN_WEIGHTS,    MBConv6_2_project_conv_BN_weights,		sizeof(MBConv6_2_project_conv_BN_weights) / sizeof(float),
  &MBConv6_2_PRJ_BN_BIAS,				MBConv6_2_project_conv_BN_bias, 		  sizeof(MBConv6_2_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_3_EXPD_BN_MEAN,      MBConv6_3_expansion_conv_BN_mean, 		sizeof(MBConv6_3_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_3_EXPD_BN_VARIANCE,	MBConv6_3_expansion_conv_BN_variance,	sizeof(MBConv6_3_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_3_EXPD_BN_WEIGHTS,   MBConv6_3_expansion_conv_BN_weights,	sizeof(MBConv6_3_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_3_EXPD_BN_BIAS,			MBConv6_3_expansion_conv_BN_bias,		  sizeof(MBConv6_3_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_3_DW_BN_MEAN,        MBConv6_3_depthwise_conv_BN_mean,		  sizeof(MBConv6_3_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_3_DW_BN_VARIANCE,		MBConv6_3_depthwise_conv_BN_variance,	sizeof(MBConv6_3_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_3_DW_BN_WEIGHTS,     MBConv6_3_depthwise_conv_BN_weights,	sizeof(MBConv6_3_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_3_DW_BN_BIAS,				MBConv6_3_depthwise_conv_BN_bias,		  sizeof(MBConv6_3_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_3_PRJ_BN_MEAN,       MBConv6_3_project_conv_BN_mean,			  sizeof(MBConv6_3_project_conv_BN_mean) / sizeof(float),
  &MBConv6_3_PRJ_BN_VARIANCE,		MBConv6_3_project_conv_BN_variance,		sizeof(MBConv6_3_project_conv_BN_variance) / sizeof(float),
  &MBConv6_3_PRJ_BN_WEIGHTS,    MBConv6_3_project_conv_BN_weights,		sizeof(MBConv6_3_project_conv_BN_weights) / sizeof(float),
  &MBConv6_3_PRJ_BN_BIAS,				MBConv6_3_project_conv_BN_bias, 		  sizeof(MBConv6_3_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_4_EXPD_BN_MEAN,      MBConv6_4_expansion_conv_BN_mean, 		sizeof(MBConv6_4_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_4_EXPD_BN_VARIANCE,	MBConv6_4_expansion_conv_BN_variance,	sizeof(MBConv6_4_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_4_EXPD_BN_WEIGHTS,   MBConv6_4_expansion_conv_BN_weights,	sizeof(MBConv6_4_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_4_EXPD_BN_BIAS,			MBConv6_4_expansion_conv_BN_bias,		  sizeof(MBConv6_4_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_4_DW_BN_MEAN,        MBConv6_4_depthwise_conv_BN_mean,		  sizeof(MBConv6_4_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_4_DW_BN_VARIANCE,		MBConv6_4_depthwise_conv_BN_variance,	sizeof(MBConv6_4_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_4_DW_BN_WEIGHTS,     MBConv6_4_depthwise_conv_BN_weights,	sizeof(MBConv6_4_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_4_DW_BN_BIAS,				MBConv6_4_depthwise_conv_BN_bias,		  sizeof(MBConv6_4_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_4_PRJ_BN_MEAN,       MBConv6_4_project_conv_BN_mean,			  sizeof(MBConv6_4_project_conv_BN_mean) / sizeof(float),
  &MBConv6_4_PRJ_BN_VARIANCE,		MBConv6_4_project_conv_BN_variance,		sizeof(MBConv6_4_project_conv_BN_variance) / sizeof(float),
  &MBConv6_4_PRJ_BN_WEIGHTS,    MBConv6_4_project_conv_BN_weights,		sizeof(MBConv6_4_project_conv_BN_weights) / sizeof(float),
  &MBConv6_4_PRJ_BN_BIAS,				MBConv6_4_project_conv_BN_bias, 		  sizeof(MBConv6_4_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_5_EXPD_BN_MEAN,      MBConv6_5_expansion_conv_BN_mean,		  sizeof(MBConv6_5_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_5_EXPD_BN_VARIANCE,	MBConv6_5_expansion_conv_BN_variance,	sizeof(MBConv6_5_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_5_EXPD_BN_WEIGHTS,   MBConv6_5_expansion_conv_BN_weights,	sizeof(MBConv6_5_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_5_EXPD_BN_BIAS,			MBConv6_5_expansion_conv_BN_bias,		  sizeof(MBConv6_5_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_5_DW_BN_MEAN,        MBConv6_5_depthwise_conv_BN_mean,		  sizeof(MBConv6_5_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_5_DW_BN_VARIANCE,		MBConv6_5_depthwise_conv_BN_variance,	sizeof(MBConv6_5_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_5_DW_BN_WEIGHTS,     MBConv6_5_depthwise_conv_BN_weights,	sizeof(MBConv6_5_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_5_DW_BN_BIAS,				MBConv6_5_depthwise_conv_BN_bias,		  sizeof(MBConv6_5_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_5_PRJ_BN_MEAN,       MBConv6_5_project_conv_BN_mean,			  sizeof(MBConv6_5_project_conv_BN_mean) / sizeof(float),
  &MBConv6_5_PRJ_BN_VARIANCE,		MBConv6_5_project_conv_BN_variance,		sizeof(MBConv6_5_project_conv_BN_variance) / sizeof(float),
  &MBConv6_5_PRJ_BN_WEIGHTS,    MBConv6_5_project_conv_BN_weights,		sizeof(MBConv6_5_project_conv_BN_weights) / sizeof(float),
  &MBConv6_5_PRJ_BN_BIAS,				MBConv6_5_project_conv_BN_bias, 		  sizeof(MBConv6_5_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_6_EXPD_BN_MEAN,      MBConv6_6_expansion_conv_BN_mean,		  sizeof(MBConv6_6_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_6_EXPD_BN_VARIANCE,	MBConv6_6_expansion_conv_BN_variance,	sizeof(MBConv6_6_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_6_EXPD_BN_WEIGHTS,   MBConv6_6_expansion_conv_BN_weights,	sizeof(MBConv6_6_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_6_EXPD_BN_BIAS,			MBConv6_6_expansion_conv_BN_bias,		  sizeof(MBConv6_6_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_6_DW_BN_MEAN,        MBConv6_6_depthwise_conv_BN_mean,		  sizeof(MBConv6_6_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_6_DW_BN_VARIANCE,		MBConv6_6_depthwise_conv_BN_variance,	sizeof(MBConv6_6_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_6_DW_BN_WEIGHTS,     MBConv6_6_depthwise_conv_BN_weights,	sizeof(MBConv6_6_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_6_DW_BN_BIAS,				MBConv6_6_depthwise_conv_BN_bias,		  sizeof(MBConv6_6_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_6_PRJ_BN_MEAN,       MBConv6_6_project_conv_BN_mean,			  sizeof(MBConv6_6_project_conv_BN_mean) / sizeof(float),
  &MBConv6_6_PRJ_BN_VARIANCE,		MBConv6_6_project_conv_BN_variance,		sizeof(MBConv6_6_project_conv_BN_variance) / sizeof(float),
  &MBConv6_6_PRJ_BN_WEIGHTS,    MBConv6_6_project_conv_BN_weights,		sizeof(MBConv6_6_project_conv_BN_weights) / sizeof(float),
  &MBConv6_6_PRJ_BN_BIAS,				MBConv6_6_project_conv_BN_bias, 		  sizeof(MBConv6_6_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_7_EXPD_BN_MEAN,      MBConv6_7_expansion_conv_BN_mean,		  sizeof(MBConv6_7_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_7_EXPD_BN_VARIANCE,	MBConv6_7_expansion_conv_BN_variance,	sizeof(MBConv6_7_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_7_EXPD_BN_WEIGHTS,   MBConv6_7_expansion_conv_BN_weights,	sizeof(MBConv6_7_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_7_EXPD_BN_BIAS,			MBConv6_7_expansion_conv_BN_bias,		  sizeof(MBConv6_7_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_7_DW_BN_MEAN,        MBConv6_7_depthwise_conv_BN_mean,		  sizeof(MBConv6_7_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_7_DW_BN_VARIANCE,		MBConv6_7_depthwise_conv_BN_variance,	sizeof(MBConv6_7_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_7_DW_BN_WEIGHTS,     MBConv6_7_depthwise_conv_BN_weights,	sizeof(MBConv6_7_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_7_DW_BN_BIAS,				MBConv6_7_depthwise_conv_BN_bias,		  sizeof(MBConv6_7_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_7_PRJ_BN_MEAN,       MBConv6_7_project_conv_BN_mean,			  sizeof(MBConv6_7_project_conv_BN_mean) / sizeof(float),
  &MBConv6_7_PRJ_BN_VARIANCE,		MBConv6_7_project_conv_BN_variance,		sizeof(MBConv6_7_project_conv_BN_variance) / sizeof(float),
  &MBConv6_7_PRJ_BN_WEIGHTS,    MBConv6_7_project_conv_BN_weights,		sizeof(MBConv6_7_project_conv_BN_weights) / sizeof(float),
  &MBConv6_7_PRJ_BN_BIAS,				MBConv6_7_project_conv_BN_bias, 		  sizeof(MBConv6_7_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_8_EXPD_BN_MEAN,      MBConv6_8_expansion_conv_BN_mean,		  sizeof(MBConv6_8_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_8_EXPD_BN_VARIANCE,	MBConv6_8_expansion_conv_BN_variance,	sizeof(MBConv6_8_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_8_EXPD_BN_WEIGHTS,   MBConv6_8_expansion_conv_BN_weights,	sizeof(MBConv6_8_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_8_EXPD_BN_BIAS,			MBConv6_8_expansion_conv_BN_bias,		  sizeof(MBConv6_8_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_8_DW_BN_MEAN,        MBConv6_8_depthwise_conv_BN_mean,		  sizeof(MBConv6_8_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_8_DW_BN_VARIANCE,		MBConv6_8_depthwise_conv_BN_variance,	sizeof(MBConv6_8_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_8_DW_BN_WEIGHTS,     MBConv6_8_depthwise_conv_BN_weights,	sizeof(MBConv6_8_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_8_DW_BN_BIAS,				MBConv6_8_depthwise_conv_BN_bias,		  sizeof(MBConv6_8_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_8_PRJ_BN_MEAN,       MBConv6_8_project_conv_BN_mean,			  sizeof(MBConv6_8_project_conv_BN_mean) / sizeof(float),
  &MBConv6_8_PRJ_BN_VARIANCE,		MBConv6_8_project_conv_BN_variance,		sizeof(MBConv6_8_project_conv_BN_variance) / sizeof(float),
  &MBConv6_8_PRJ_BN_WEIGHTS,    MBConv6_8_project_conv_BN_weights,		sizeof(MBConv6_8_project_conv_BN_weights) / sizeof(float),
  &MBConv6_8_PRJ_BN_BIAS,				MBConv6_8_project_conv_BN_bias, 		  sizeof(MBConv6_8_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_9_EXPD_BN_MEAN,      MBConv6_9_expansion_conv_BN_mean,		  sizeof(MBConv6_9_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_9_EXPD_BN_VARIANCE,	MBConv6_9_expansion_conv_BN_variance,	sizeof(MBConv6_9_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_9_EXPD_BN_WEIGHTS,   MBConv6_9_expansion_conv_BN_weights,	sizeof(MBConv6_9_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_9_EXPD_BN_BIAS,			MBConv6_9_expansion_conv_BN_bias,		  sizeof(MBConv6_9_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_9_DW_BN_MEAN,        MBConv6_9_depthwise_conv_BN_mean,		  sizeof(MBConv6_9_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_9_DW_BN_VARIANCE,		MBConv6_9_depthwise_conv_BN_variance,	sizeof(MBConv6_9_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_9_DW_BN_WEIGHTS,     MBConv6_9_depthwise_conv_BN_weights,	sizeof(MBConv6_9_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_9_DW_BN_BIAS,				MBConv6_9_depthwise_conv_BN_bias,		  sizeof(MBConv6_9_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_9_PRJ_BN_MEAN,       MBConv6_9_project_conv_BN_mean,			  sizeof(MBConv6_9_project_conv_BN_mean) / sizeof(float),
  &MBConv6_9_PRJ_BN_VARIANCE,		MBConv6_9_project_conv_BN_variance,		sizeof(MBConv6_9_project_conv_BN_variance) / sizeof(float),
  &MBConv6_9_PRJ_BN_WEIGHTS,    MBConv6_9_project_conv_BN_weights,		sizeof(MBConv6_9_project_conv_BN_weights) / sizeof(float),
  &MBConv6_9_PRJ_BN_BIAS,				MBConv6_9_project_conv_BN_bias, 		  sizeof(MBConv6_9_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_10_EXPD_BN_MEAN,     MBConv6_10_expansion_conv_BN_mean,    sizeof(MBConv6_10_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_10_EXPD_BN_VARIANCE,	MBConv6_10_expansion_conv_BN_variance,sizeof(MBConv6_10_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_10_EXPD_BN_WEIGHTS,  MBConv6_10_expansion_conv_BN_weights,	sizeof(MBConv6_10_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_10_EXPD_BN_BIAS,			MBConv6_10_expansion_conv_BN_bias,		sizeof(MBConv6_10_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_10_DW_BN_MEAN,       MBConv6_10_depthwise_conv_BN_mean,		sizeof(MBConv6_10_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_10_DW_BN_VARIANCE,		MBConv6_10_depthwise_conv_BN_variance,sizeof(MBConv6_10_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_10_DW_BN_WEIGHTS,    MBConv6_10_depthwise_conv_BN_weights,	sizeof(MBConv6_10_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_10_DW_BN_BIAS,				MBConv6_10_depthwise_conv_BN_bias,		sizeof(MBConv6_10_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_10_PRJ_BN_MEAN,      MBConv6_10_project_conv_BN_mean,		  sizeof(MBConv6_10_project_conv_BN_mean) / sizeof(float),
  &MBConv6_10_PRJ_BN_VARIANCE,	MBConv6_10_project_conv_BN_variance,	sizeof(MBConv6_10_project_conv_BN_variance) / sizeof(float),
  &MBConv6_10_PRJ_BN_WEIGHTS,   MBConv6_10_project_conv_BN_weights,		sizeof(MBConv6_10_project_conv_BN_weights) / sizeof(float),
  &MBConv6_10_PRJ_BN_BIAS,			MBConv6_10_project_conv_BN_bias, 		  sizeof(MBConv6_10_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_11_EXPD_BN_MEAN,     MBConv6_11_expansion_conv_BN_mean,		sizeof(MBConv6_11_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_11_EXPD_BN_VARIANCE,	MBConv6_11_expansion_conv_BN_variance,sizeof(MBConv6_11_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_11_EXPD_BN_WEIGHTS,  MBConv6_11_expansion_conv_BN_weights,	sizeof(MBConv6_11_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_11_EXPD_BN_BIAS,			MBConv6_11_expansion_conv_BN_bias,		sizeof(MBConv6_11_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_11_DW_BN_MEAN,       MBConv6_11_depthwise_conv_BN_mean,		sizeof(MBConv6_11_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_11_DW_BN_VARIANCE,		MBConv6_11_depthwise_conv_BN_variance,sizeof(MBConv6_11_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_11_DW_BN_WEIGHTS,    MBConv6_11_depthwise_conv_BN_weights,	sizeof(MBConv6_11_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_11_DW_BN_BIAS,				MBConv6_11_depthwise_conv_BN_bias,		sizeof(MBConv6_11_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_11_PRJ_BN_MEAN,      MBConv6_11_project_conv_BN_mean,		  sizeof(MBConv6_11_project_conv_BN_mean) / sizeof(float),
  &MBConv6_11_PRJ_BN_VARIANCE,	MBConv6_11_project_conv_BN_variance,	sizeof(MBConv6_11_project_conv_BN_variance) / sizeof(float),
  &MBConv6_11_PRJ_BN_WEIGHTS,   MBConv6_11_project_conv_BN_weights,		sizeof(MBConv6_11_project_conv_BN_weights) / sizeof(float),
  &MBConv6_11_PRJ_BN_BIAS,			MBConv6_11_project_conv_BN_bias, 		  sizeof(MBConv6_11_project_conv_BN_bias) / sizeof(float));

  DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_12_EXPD_BN_MEAN,     MBConv6_12_expansion_conv_BN_mean,		sizeof(MBConv6_12_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_12_EXPD_BN_VARIANCE,	MBConv6_12_expansion_conv_BN_variance,sizeof(MBConv6_12_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_12_EXPD_BN_WEIGHTS,  MBConv6_12_expansion_conv_BN_weights,	sizeof(MBConv6_12_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_12_EXPD_BN_BIAS,			MBConv6_12_expansion_conv_BN_bias,		sizeof(MBConv6_12_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_12_DW_BN_MEAN,       MBConv6_12_depthwise_conv_BN_mean,		sizeof(MBConv6_12_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_12_DW_BN_VARIANCE,		MBConv6_12_depthwise_conv_BN_variance,sizeof(MBConv6_12_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_12_DW_BN_WEIGHTS,    MBConv6_12_depthwise_conv_BN_weights,	sizeof(MBConv6_12_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_12_DW_BN_BIAS,				MBConv6_12_depthwise_conv_BN_bias,		sizeof(MBConv6_12_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_12_PRJ_BN_MEAN,      MBConv6_12_project_conv_BN_mean,		  sizeof(MBConv6_12_project_conv_BN_mean) / sizeof(float),
  &MBConv6_12_PRJ_BN_VARIANCE,	MBConv6_12_project_conv_BN_variance,	sizeof(MBConv6_12_project_conv_BN_variance) / sizeof(float),
  &MBConv6_12_PRJ_BN_WEIGHTS,   MBConv6_12_project_conv_BN_weights,		sizeof(MBConv6_12_project_conv_BN_weights) / sizeof(float),
  &MBConv6_12_PRJ_BN_BIAS,			MBConv6_12_project_conv_BN_bias, 		  sizeof(MBConv6_12_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_13_EXPD_BN_MEAN,     MBConv6_13_expansion_conv_BN_mean,		sizeof(MBConv6_13_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_13_EXPD_BN_VARIANCE,	MBConv6_13_expansion_conv_BN_variance,sizeof(MBConv6_13_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_13_EXPD_BN_WEIGHTS,  MBConv6_13_expansion_conv_BN_weights,	sizeof(MBConv6_13_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_13_EXPD_BN_BIAS,			MBConv6_13_expansion_conv_BN_bias,		sizeof(MBConv6_13_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_13_DW_BN_MEAN,       MBConv6_13_depthwise_conv_BN_mean,		sizeof(MBConv6_13_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_13_DW_BN_VARIANCE,		MBConv6_13_depthwise_conv_BN_variance,sizeof(MBConv6_13_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_13_DW_BN_WEIGHTS,    MBConv6_13_depthwise_conv_BN_weights,	sizeof(MBConv6_13_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_13_DW_BN_BIAS,				MBConv6_13_depthwise_conv_BN_bias,		sizeof(MBConv6_13_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_13_PRJ_BN_MEAN,      MBConv6_13_project_conv_BN_mean,		  sizeof(MBConv6_13_project_conv_BN_mean) / sizeof(float),
  &MBConv6_13_PRJ_BN_VARIANCE,	MBConv6_13_project_conv_BN_variance,	sizeof(MBConv6_13_project_conv_BN_variance) / sizeof(float),
  &MBConv6_13_PRJ_BN_WEIGHTS,   MBConv6_13_project_conv_BN_weights,		sizeof(MBConv6_13_project_conv_BN_weights) / sizeof(float),
  &MBConv6_13_PRJ_BN_BIAS,			MBConv6_13_project_conv_BN_bias, 		  sizeof(MBConv6_13_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_14_EXPD_BN_MEAN,     MBConv6_14_expansion_conv_BN_mean,		sizeof(MBConv6_14_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_14_EXPD_BN_VARIANCE,	MBConv6_14_expansion_conv_BN_variance,sizeof(MBConv6_14_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_14_EXPD_BN_WEIGHTS,  MBConv6_14_expansion_conv_BN_weights,	sizeof(MBConv6_14_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_14_EXPD_BN_BIAS,			MBConv6_14_expansion_conv_BN_bias,		sizeof(MBConv6_14_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_14_DW_BN_MEAN,       MBConv6_14_depthwise_conv_BN_mean,		sizeof(MBConv6_14_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_14_DW_BN_VARIANCE,		MBConv6_14_depthwise_conv_BN_variance,sizeof(MBConv6_14_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_14_DW_BN_WEIGHTS,    MBConv6_14_depthwise_conv_BN_weights,	sizeof(MBConv6_14_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_14_DW_BN_BIAS,				MBConv6_14_depthwise_conv_BN_bias,		sizeof(MBConv6_14_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_14_PRJ_BN_MEAN,      MBConv6_14_project_conv_BN_mean,		  sizeof(MBConv6_14_project_conv_BN_mean) / sizeof(float),
  &MBConv6_14_PRJ_BN_VARIANCE,	MBConv6_14_project_conv_BN_variance,	sizeof(MBConv6_14_project_conv_BN_variance) / sizeof(float),
  &MBConv6_14_PRJ_BN_WEIGHTS,   MBConv6_14_project_conv_BN_weights,		sizeof(MBConv6_14_project_conv_BN_weights) / sizeof(float),
  &MBConv6_14_PRJ_BN_BIAS,			MBConv6_14_project_conv_BN_bias, 		  sizeof(MBConv6_14_project_conv_BN_bias) / sizeof(float));

DEFINE_FILTERS_FOR_MBCONV_BN(	
  &MBConv6_15_EXPD_BN_MEAN,     MBConv6_15_expansion_conv_BN_mean,		sizeof(MBConv6_15_expansion_conv_BN_mean) / sizeof(float),
  &MBConv6_15_EXPD_BN_VARIANCE,	MBConv6_15_expansion_conv_BN_variance,sizeof(MBConv6_15_expansion_conv_BN_variance) / sizeof(float),
  &MBConv6_15_EXPD_BN_WEIGHTS,  MBConv6_15_expansion_conv_BN_weights,	sizeof(MBConv6_15_expansion_conv_BN_weights) / sizeof(float),
  &MBConv6_15_EXPD_BN_BIAS,			MBConv6_15_expansion_conv_BN_bias,		sizeof(MBConv6_15_expansion_conv_BN_bias) / sizeof(float),
  &MBConv6_15_DW_BN_MEAN,       MBConv6_15_depthwise_conv_BN_mean,		sizeof(MBConv6_15_depthwise_conv_BN_mean) / sizeof(float),
  &MBConv6_15_DW_BN_VARIANCE,		MBConv6_15_depthwise_conv_BN_variance,sizeof(MBConv6_15_depthwise_conv_BN_variance) / sizeof(float),	
  &MBConv6_15_DW_BN_WEIGHTS,    MBConv6_15_depthwise_conv_BN_weights,	sizeof(MBConv6_15_depthwise_conv_BN_weights) / sizeof(float),
  &MBConv6_15_DW_BN_BIAS,				MBConv6_15_depthwise_conv_BN_bias,		sizeof(MBConv6_15_depthwise_conv_BN_bias) / sizeof(float),
  &MBConv6_15_PRJ_BN_MEAN,      MBConv6_15_project_conv_BN_mean,		  sizeof(MBConv6_15_project_conv_BN_mean) / sizeof(float),
  &MBConv6_15_PRJ_BN_VARIANCE,	MBConv6_15_project_conv_BN_variance,	sizeof(MBConv6_15_project_conv_BN_variance) / sizeof(float),
  &MBConv6_15_PRJ_BN_WEIGHTS,   MBConv6_15_project_conv_BN_weights,		sizeof(MBConv6_15_project_conv_BN_weights) / sizeof(float),
  &MBConv6_15_PRJ_BN_BIAS,			MBConv6_15_project_conv_BN_bias, 		  sizeof(MBConv6_15_project_conv_BN_bias) / sizeof(float));


set_allocate_copy_array_Device(&D_STEM_BN_MEAN, Stem_BN_mean,
                sizeof(Stem_BN_mean)/sizeof(float), 1, 1,
                "STEM MEAN"); 
set_allocate_copy_array_Device(&D_STEM_BN_VARIANCE, Stem_BN_variance,
                sizeof(Stem_BN_variance)/sizeof(float), 1, 1,
                "STEAM VARIANCE"); 
set_allocate_copy_array_Device(&D_STEM_BN_WEIGHTS, Stem_BN_weights,
                sizeof(Stem_BN_weights)/sizeof(float), 1, 1,
                "STEM WEIGHTS"); 
set_allocate_copy_array_Device(&D_STEM_BN_BIAS, Stem_BN_bias,
                sizeof(Stem_BN_bias)/sizeof(float), 1, 1,
                "STEM BIAS"); 
                
set_allocate_copy_array_Device(&D_HEAD_BN_MEAN, Head_BN_mean,
                sizeof(Head_BN_mean)/sizeof(float), 1, 1,
                "HEAD MEAN"); 
set_allocate_copy_array_Device(&D_HEAD_BN_VARIANCE, Head_BN_variance,
                sizeof(Head_BN_variance)/sizeof(float), 1, 1,
                "HEAD VARIANCE"); 
set_allocate_copy_array_Device(&D_HEAD_BN_WEIGHTS, Head_BN_weights,
                sizeof(Head_BN_weights)/sizeof(float), 1, 1,
                "HEAD WEIGHTS"); 
set_allocate_copy_array_Device(&D_HEAD_BN_BIAS, Head_BN_bias,
                sizeof(Head_BN_bias)/sizeof(float), 1, 1,
                "HEAD BIAS"); 
start();
  // 3. Move through all layers starting from stem layer till head layer
  Matrix ConvOutStem;
  STEM_LAYER(&DInput_Mat, &F_STEM,
              INPUT_IMAGE_HEIGHT, INPUT_IMAGE_WIDTH, INPUT_IMAGE_DEPTH,
              STEM_FILTER_HEIGHT, STEM_FILTER_WIDTH, STEM_FILTER_DEPTH, 
              STEM_FILTER_DENSITY,STEM_PADDING,      STEM_STRIDE,
              &ConvOutStem);


  Matrix ConvOut1_0;
  MBCONV1_0_flag = 1;  

  MBConv_Layer(&ConvOutStem, &ConvOut1_0,
                &D_MBConv_1_0_EXPD_WEIGHTS, &D_MBConv_1_0_DW_WEIGHTS,
                &D_MBConv_1_0_SQZ_1_WEIGHTS,&D_MBConv_1_0_SQZ_2_WEIGHTS,
                &D_MBConv_1_0_PRJ_WEIGHTS,
                MBCONV_1_0_EXPD_F_DENSITY,  MBCONV_1_0_DW_F_DENSITY, 
                MBCONV_1_0_SQZ_1_F_DENSITY, MBCONV_1_0_SQZ_2_F_DENSITY, 
                MBCONV_1_0_PRJ_F_DENSITY,
                ConvOutStem.depth,          MBCONV_1_0_PRJ_F_DENSITY, MBCONV_1_0_DW_F_HEIGHT,
                MBCONV_1_0_STRIDE,          MBCONV_1_0_PADDING, MBCONV_1_0_SKIP,
                &MBConv1_0_SQZ_1_bias, 	    &MBConv1_0_SQZ_2_bias,
                NULL,                       NULL,
                NULL,                       NULL,
                &MBConv1_0_DW_BN_MEAN,      &MBConv1_0_DW_BN_VARIANCE,
                &MBConv1_0_DW_BN_WEIGHTS,   &MBConv1_0_DW_BN_BIAS,
                &MBConv1_0_PRJ_BN_MEAN,     &MBConv1_0_PRJ_BN_VARIANCE,
                &MBConv1_0_PRJ_BN_WEIGHTS,  &MBConv1_0_PRJ_BN_BIAS);
  MBCONV1_0_flag = 0;
  

  Matrix ConvOut;
  MBConv_Layer(&ConvOut1_0, &ConvOut,
                &D_MBConv_6_1_EXPD_WEIGHTS, &D_MBConv_6_1_DW_WEIGHTS,
                &D_MBConv_6_1_SQZ_1_WEIGHTS,&D_MBConv_6_1_SQZ_2_WEIGHTS,
                &D_MBConv_6_1_PRJ_WEIGHTS,
                MBCONV_6_1_EXPD_F_DENSITY,  MBCONV_6_1_DW_F_DENSITY, 
                MBCONV_6_1_SQZ_1_F_DENSITY, MBCONV_6_1_SQZ_2_F_DENSITY, 
                MBCONV_6_1_PRJ_F_DENSITY,
                ConvOut1_0.depth,           MBCONV_6_1_PRJ_F_DENSITY, MBCONV_6_1_DW_F_HEIGHT,
                MBCONV_6_1_STRIDE,          MBCONV_6_1_PADDING, MBCONV_6_1_SKIP,
                &MBConv6_1_SQZ_1_bias, 	    &MBConv6_1_SQZ_2_bias,
                &MBConv6_1_EXPD_BN_MEAN,    &MBConv6_1_EXPD_BN_VARIANCE,
                &MBConv6_1_EXPD_BN_WEIGHTS, &MBConv6_1_EXPD_BN_BIAS,
                &MBConv6_1_DW_BN_MEAN,      &MBConv6_1_DW_BN_VARIANCE,
                &MBConv6_1_DW_BN_WEIGHTS,   &MBConv6_1_DW_BN_BIAS,
                &MBConv6_1_PRJ_BN_MEAN,     &MBConv6_1_PRJ_BN_VARIANCE,
                &MBConv6_1_PRJ_BN_WEIGHTS,  &MBConv6_1_PRJ_BN_BIAS);


  Matrix ConvOut2;
  MBConv_Layer(&ConvOut, &ConvOut2,
                &D_MBConv_6_2_EXPD_WEIGHTS, &D_MBConv_6_2_DW_WEIGHTS,
                &D_MBConv_6_2_SQZ_1_WEIGHTS,&D_MBConv_6_2_SQZ_2_WEIGHTS,
                &D_MBConv_6_2_PRJ_WEIGHTS,
                MBCONV_6_2_EXPD_F_DENSITY,  MBCONV_6_2_DW_F_DENSITY, 
                MBCONV_6_2_SQZ_1_F_DENSITY, MBCONV_6_2_SQZ_2_F_DENSITY, 
                MBCONV_6_2_PRJ_F_DENSITY,
                ConvOut.depth,              MBCONV_6_2_PRJ_F_DENSITY, MBCONV_6_2_DW_F_HEIGHT,
                MBCONV_6_2_STRIDE,          MBCONV_6_2_PADDING, MBCONV_6_2_SKIP,
                &MBConv6_2_SQZ_1_bias, 	    &MBConv6_2_SQZ_2_bias,
                &MBConv6_2_EXPD_BN_MEAN,    &MBConv6_2_EXPD_BN_VARIANCE,
                &MBConv6_2_EXPD_BN_WEIGHTS, &MBConv6_2_EXPD_BN_BIAS,
                &MBConv6_2_DW_BN_MEAN,      &MBConv6_2_DW_BN_VARIANCE,
                &MBConv6_2_DW_BN_WEIGHTS,   &MBConv6_2_DW_BN_BIAS,
                &MBConv6_2_PRJ_BN_MEAN,     &MBConv6_2_PRJ_BN_VARIANCE,
                &MBConv6_2_PRJ_BN_WEIGHTS,  &MBConv6_2_PRJ_BN_BIAS); 


  Matrix ConvOut3;
	MBConv_Layer(&ConvOut2, &ConvOut3,
                &D_MBConv_6_3_EXPD_WEIGHTS, &D_MBConv_6_3_DW_WEIGHTS,
                &D_MBConv_6_3_SQZ_1_WEIGHTS,&D_MBConv_6_3_SQZ_2_WEIGHTS,
                &D_MBConv_6_3_PRJ_WEIGHTS,
                MBCONV_6_3_EXPD_F_DENSITY,  MBCONV_6_3_DW_F_DENSITY, 
                MBCONV_6_3_SQZ_1_F_DENSITY, MBCONV_6_3_SQZ_2_F_DENSITY, 
                MBCONV_6_3_PRJ_F_DENSITY,
                ConvOut2.depth,             MBCONV_6_3_PRJ_F_DENSITY, MBCONV_6_3_DW_F_HEIGHT,
                MBCONV_6_3_STRIDE,          MBCONV_6_3_PADDING, MBCONV_6_3_SKIP,
                &MBConv6_3_SQZ_1_bias,  	  &MBConv6_3_SQZ_2_bias,
                &MBConv6_3_EXPD_BN_MEAN,    &MBConv6_3_EXPD_BN_VARIANCE,
                &MBConv6_3_EXPD_BN_WEIGHTS, &MBConv6_3_EXPD_BN_BIAS,
                &MBConv6_3_DW_BN_MEAN,      &MBConv6_3_DW_BN_VARIANCE,
                &MBConv6_3_DW_BN_WEIGHTS,   &MBConv6_3_DW_BN_BIAS,
                &MBConv6_3_PRJ_BN_MEAN,     &MBConv6_3_PRJ_BN_VARIANCE,
                &MBConv6_3_PRJ_BN_WEIGHTS,  &MBConv6_3_PRJ_BN_BIAS);  
 

  // MBConv6_4 layer implementation

  Matrix ConvOut4;
  MBConv_Layer(&ConvOut3, &ConvOut4,
                &D_MBConv_6_4_EXPD_WEIGHTS, &D_MBConv_6_4_DW_WEIGHTS,
                &D_MBConv_6_4_SQZ_1_WEIGHTS,&D_MBConv_6_4_SQZ_2_WEIGHTS,
                &D_MBConv_6_4_PRJ_WEIGHTS,
                MBCONV_6_4_EXPD_F_DENSITY,  MBCONV_6_4_DW_F_DENSITY, 
                MBCONV_6_4_SQZ_1_F_DENSITY, MBCONV_6_4_SQZ_2_F_DENSITY, 
                MBCONV_6_4_PRJ_F_DENSITY,
                ConvOut3.depth,             MBCONV_6_4_PRJ_F_DENSITY, MBCONV_6_4_DW_F_HEIGHT,
                MBCONV_6_4_STRIDE,          MBCONV_6_4_PADDING, MBCONV_6_4_SKIP,
                &MBConv6_4_SQZ_1_bias,  	  &MBConv6_4_SQZ_2_bias,
                &MBConv6_4_EXPD_BN_MEAN,    &MBConv6_4_EXPD_BN_VARIANCE,
                &MBConv6_4_EXPD_BN_WEIGHTS, &MBConv6_4_EXPD_BN_BIAS,
                &MBConv6_4_DW_BN_MEAN,      &MBConv6_4_DW_BN_VARIANCE,
                &MBConv6_4_DW_BN_WEIGHTS,   &MBConv6_4_DW_BN_BIAS,
                &MBConv6_4_PRJ_BN_MEAN,     &MBConv6_4_PRJ_BN_VARIANCE,
                &MBConv6_4_PRJ_BN_WEIGHTS,  &MBConv6_4_PRJ_BN_BIAS);   
  

  Matrix ConvOut5;
  MBConv_Layer(&ConvOut4, &ConvOut5,
                &D_MBConv_6_5_EXPD_WEIGHTS, &D_MBConv_6_5_DW_WEIGHTS,
                &D_MBConv_6_5_SQZ_1_WEIGHTS,&D_MBConv_6_5_SQZ_2_WEIGHTS,
                &D_MBConv_6_5_PRJ_WEIGHTS,
                MBCONV_6_5_EXPD_F_DENSITY,  MBCONV_6_5_DW_F_DENSITY, 
                MBCONV_6_5_SQZ_1_F_DENSITY, MBCONV_6_5_SQZ_2_F_DENSITY, 
                MBCONV_6_5_PRJ_F_DENSITY,
                ConvOut4.depth,             MBCONV_6_5_PRJ_F_DENSITY, MBCONV_6_5_DW_F_HEIGHT,
                MBCONV_6_5_STRIDE,          MBCONV_6_5_PADDING, MBCONV_6_5_SKIP,
                &MBConv6_5_SQZ_1_bias,  	  &MBConv6_5_SQZ_2_bias,
                &MBConv6_5_EXPD_BN_MEAN,    &MBConv6_5_EXPD_BN_VARIANCE,
                &MBConv6_5_EXPD_BN_WEIGHTS, &MBConv6_5_EXPD_BN_BIAS,
                &MBConv6_5_DW_BN_MEAN,      &MBConv6_5_DW_BN_VARIANCE,
                &MBConv6_5_DW_BN_WEIGHTS,   &MBConv6_5_DW_BN_BIAS,
                &MBConv6_5_PRJ_BN_MEAN,     &MBConv6_5_PRJ_BN_VARIANCE,
                &MBConv6_5_PRJ_BN_WEIGHTS,  &MBConv6_5_PRJ_BN_BIAS); 
            


  // MBConv6_6 layer implementation


  Matrix ConvOut6;
  MBConv_Layer(&ConvOut5, &ConvOut6,
                &D_MBConv_6_6_EXPD_WEIGHTS, &D_MBConv_6_6_DW_WEIGHTS,
                &D_MBConv_6_6_SQZ_1_WEIGHTS,&D_MBConv_6_6_SQZ_2_WEIGHTS,
                &D_MBConv_6_6_PRJ_WEIGHTS,
                MBCONV_6_6_EXPD_F_DENSITY,  MBCONV_6_6_DW_F_DENSITY, 
                MBCONV_6_6_SQZ_1_F_DENSITY, MBCONV_6_6_SQZ_2_F_DENSITY, 
                MBCONV_6_6_PRJ_F_DENSITY,
                ConvOut5.depth,             MBCONV_6_6_PRJ_F_DENSITY, MBCONV_6_6_DW_F_HEIGHT,
                MBCONV_6_6_STRIDE,          MBCONV_6_6_PADDING, MBCONV_6_6_SKIP,
                &MBConv6_6_SQZ_1_bias, 	    &MBConv6_6_SQZ_2_bias,
                &MBConv6_6_EXPD_BN_MEAN,    &MBConv6_6_EXPD_BN_VARIANCE,
                &MBConv6_6_EXPD_BN_WEIGHTS, &MBConv6_6_EXPD_BN_BIAS,
                &MBConv6_6_DW_BN_MEAN,      &MBConv6_6_DW_BN_VARIANCE,
                &MBConv6_6_DW_BN_WEIGHTS,   &MBConv6_6_DW_BN_BIAS,
                &MBConv6_6_PRJ_BN_MEAN,     &MBConv6_6_PRJ_BN_VARIANCE,
                &MBConv6_6_PRJ_BN_WEIGHTS,  &MBConv6_6_PRJ_BN_BIAS);  
            


  // MBConv6_7 layer implementation


  Matrix ConvOut7;
  MBConv_Layer(&ConvOut6, &ConvOut7,
                &D_MBConv_6_7_EXPD_WEIGHTS, &D_MBConv_6_7_DW_WEIGHTS,
                &D_MBConv_6_7_SQZ_1_WEIGHTS,&D_MBConv_6_7_SQZ_2_WEIGHTS,
                &D_MBConv_6_7_PRJ_WEIGHTS,
                MBCONV_6_7_EXPD_F_DENSITY,  MBCONV_6_7_DW_F_DENSITY, 
                MBCONV_6_7_SQZ_1_F_DENSITY, MBCONV_6_7_SQZ_2_F_DENSITY, 
                MBCONV_6_7_PRJ_F_DENSITY,
                ConvOut6.depth,             MBCONV_6_7_PRJ_F_DENSITY, MBCONV_6_7_DW_F_HEIGHT,                   
                MBCONV_6_7_STRIDE,          MBCONV_6_7_PADDING, MBCONV_6_7_SKIP,
                &MBConv6_7_SQZ_1_bias,  	  &MBConv6_7_SQZ_2_bias,
                &MBConv6_7_EXPD_BN_MEAN,    &MBConv6_7_EXPD_BN_VARIANCE,
                &MBConv6_7_EXPD_BN_WEIGHTS, &MBConv6_7_EXPD_BN_BIAS,
                &MBConv6_7_DW_BN_MEAN,      &MBConv6_7_DW_BN_VARIANCE,
                &MBConv6_7_DW_BN_WEIGHTS,   &MBConv6_7_DW_BN_BIAS,
                &MBConv6_7_PRJ_BN_MEAN,     &MBConv6_7_PRJ_BN_VARIANCE,
                &MBConv6_7_PRJ_BN_WEIGHTS,  &MBConv6_7_PRJ_BN_BIAS);  
          


  // MBConv6_8 layer implementation
  Matrix ConvOut8;
  MBConv_Layer(&ConvOut7, &ConvOut8,
                &D_MBConv_6_8_EXPD_WEIGHTS, &D_MBConv_6_8_DW_WEIGHTS,
                &D_MBConv_6_8_SQZ_1_WEIGHTS,&D_MBConv_6_8_SQZ_2_WEIGHTS,
                &D_MBConv_6_8_PRJ_WEIGHTS,
                MBCONV_6_8_EXPD_F_DENSITY,  MBCONV_6_8_DW_F_DENSITY, 
                MBCONV_6_8_SQZ_1_F_DENSITY, MBCONV_6_8_SQZ_2_F_DENSITY, 
                MBCONV_6_8_PRJ_F_DENSITY,
                ConvOut7.depth,             MBCONV_6_8_PRJ_F_DENSITY, MBCONV_6_8_DW_F_HEIGHT,    
                MBCONV_6_8_STRIDE,          MBCONV_6_8_PADDING, MBCONV_6_8_SKIP,
                &MBConv6_8_SQZ_1_bias,      &MBConv6_8_SQZ_2_bias,
                &MBConv6_8_EXPD_BN_MEAN,    &MBConv6_8_EXPD_BN_VARIANCE,
                &MBConv6_8_EXPD_BN_WEIGHTS, &MBConv6_8_EXPD_BN_BIAS,
                &MBConv6_8_DW_BN_MEAN,      &MBConv6_8_DW_BN_VARIANCE,
                &MBConv6_8_DW_BN_WEIGHTS,   &MBConv6_8_DW_BN_BIAS,
                &MBConv6_8_PRJ_BN_MEAN,     &MBConv6_8_PRJ_BN_VARIANCE,
                &MBConv6_8_PRJ_BN_WEIGHTS,  &MBConv6_8_PRJ_BN_BIAS); 
        


  // MBConv6_9 layer implementation
  Matrix ConvOut9;
  MBConv_Layer(&ConvOut8, &ConvOut9,
                &D_MBConv_6_9_EXPD_WEIGHTS, &D_MBConv_6_9_DW_WEIGHTS,
                &D_MBConv_6_9_SQZ_1_WEIGHTS,&D_MBConv_6_9_SQZ_2_WEIGHTS,
                &D_MBConv_6_9_PRJ_WEIGHTS,
                MBCONV_6_9_EXPD_F_DENSITY,  MBCONV_6_9_DW_F_DENSITY, 
                MBCONV_6_9_SQZ_1_F_DENSITY, MBCONV_6_9_SQZ_2_F_DENSITY, 
                MBCONV_6_9_PRJ_F_DENSITY,
                ConvOut8.depth,             MBCONV_6_9_PRJ_F_DENSITY, MBCONV_6_9_DW_F_HEIGHT,
                MBCONV_6_9_STRIDE,          MBCONV_6_9_PADDING, MBCONV_6_9_SKIP,
                &MBConv6_9_SQZ_1_bias,  	  &MBConv6_9_SQZ_2_bias,
                &MBConv6_9_EXPD_BN_MEAN,    &MBConv6_9_EXPD_BN_VARIANCE,
                &MBConv6_9_EXPD_BN_WEIGHTS, &MBConv6_9_EXPD_BN_BIAS,
                &MBConv6_9_DW_BN_MEAN,      &MBConv6_9_DW_BN_VARIANCE,
                &MBConv6_9_DW_BN_WEIGHTS,   &MBConv6_9_DW_BN_BIAS,
                &MBConv6_9_PRJ_BN_MEAN,     &MBConv6_9_PRJ_BN_VARIANCE,
                &MBConv6_9_PRJ_BN_WEIGHTS,  &MBConv6_9_PRJ_BN_BIAS);  				  



  // MBConv6_10 layer implementation
  Matrix ConvOut10;
	MBConv_Layer(&ConvOut9, &ConvOut10,
                &D_MBConv_6_10_EXPD_WEIGHTS,  &D_MBConv_6_10_DW_WEIGHTS,
                &D_MBConv_6_10_SQZ_1_WEIGHTS, &D_MBConv_6_10_SQZ_2_WEIGHTS,
                &D_MBConv_6_10_PRJ_WEIGHTS,
                MBCONV_6_10_EXPD_F_DENSITY,   MBCONV_6_10_DW_F_DENSITY, 
                MBCONV_6_10_SQZ_1_F_DENSITY,  MBCONV_6_10_SQZ_2_F_DENSITY, 
                MBCONV_6_10_PRJ_F_DENSITY,
                ConvOut9.depth,               MBCONV_6_10_PRJ_F_DENSITY, MBCONV_6_10_DW_F_HEIGHT,
                MBCONV_6_10_STRIDE,           MBCONV_6_10_PADDING, MBCONV_6_10_SKIP,
                &MBConv6_10_SQZ_1_bias, 	    &MBConv6_10_SQZ_2_bias,
                &MBConv6_10_EXPD_BN_MEAN,     &MBConv6_10_EXPD_BN_VARIANCE,
                &MBConv6_10_EXPD_BN_WEIGHTS,  &MBConv6_10_EXPD_BN_BIAS,
                &MBConv6_10_DW_BN_MEAN,       &MBConv6_10_DW_BN_VARIANCE,
                &MBConv6_10_DW_BN_WEIGHTS,    &MBConv6_10_DW_BN_BIAS,
                &MBConv6_10_PRJ_BN_MEAN,      &MBConv6_10_PRJ_BN_VARIANCE,
                &MBConv6_10_PRJ_BN_WEIGHTS,   &MBConv6_10_PRJ_BN_BIAS);   
  


  // MBConv6_11 layer implementation


  Matrix ConvOut11;
  MBConv_Layer(&ConvOut10, &ConvOut11,
                &D_MBConv_6_11_EXPD_WEIGHTS,  &D_MBConv_6_11_DW_WEIGHTS,
                &D_MBConv_6_11_SQZ_1_WEIGHTS, &D_MBConv_6_11_SQZ_2_WEIGHTS,
                &D_MBConv_6_11_PRJ_WEIGHTS,
                MBCONV_6_11_EXPD_F_DENSITY,   MBCONV_6_11_DW_F_DENSITY, 
                MBCONV_6_11_SQZ_1_F_DENSITY,  MBCONV_6_11_SQZ_2_F_DENSITY, 
                MBCONV_6_11_PRJ_F_DENSITY,  
                ConvOut10.depth,              MBCONV_6_11_PRJ_F_DENSITY, MBCONV_6_11_DW_F_HEIGHT,
                MBCONV_6_11_STRIDE,           MBCONV_6_11_PADDING, MBCONV_6_11_SKIP,
                &MBConv6_11_SQZ_1_bias,       &MBConv6_11_SQZ_2_bias,
                &MBConv6_11_EXPD_BN_MEAN,     &MBConv6_11_EXPD_BN_VARIANCE,
                &MBConv6_11_EXPD_BN_WEIGHTS,  &MBConv6_11_EXPD_BN_BIAS,
                &MBConv6_11_DW_BN_MEAN,       &MBConv6_11_DW_BN_VARIANCE,
                &MBConv6_11_DW_BN_WEIGHTS,    &MBConv6_11_DW_BN_BIAS,
                &MBConv6_11_PRJ_BN_MEAN,      &MBConv6_11_PRJ_BN_VARIANCE,
                &MBConv6_11_PRJ_BN_WEIGHTS,   &MBConv6_11_PRJ_BN_BIAS);  
  


  // MBConv6_12 layer implementation


  Matrix ConvOut12;
  MBConv_Layer(&ConvOut11, &ConvOut12,
                &D_MBConv_6_12_EXPD_WEIGHTS,  &D_MBConv_6_12_DW_WEIGHTS,
                &D_MBConv_6_12_SQZ_1_WEIGHTS, &D_MBConv_6_12_SQZ_2_WEIGHTS,
                &D_MBConv_6_12_PRJ_WEIGHTS,
                MBCONV_6_12_EXPD_F_DENSITY,   MBCONV_6_12_DW_F_DENSITY, 
                MBCONV_6_12_SQZ_1_F_DENSITY,  MBCONV_6_12_SQZ_2_F_DENSITY, 
                MBCONV_6_12_PRJ_F_DENSITY,
                ConvOut11.depth,              MBCONV_6_12_PRJ_F_DENSITY, MBCONV_6_12_DW_F_HEIGHT,
                MBCONV_6_12_STRIDE,           MBCONV_6_12_PADDING, MBCONV_6_12_SKIP,
                &MBConv6_12_SQZ_1_bias,       &MBConv6_12_SQZ_2_bias,
                &MBConv6_12_EXPD_BN_MEAN,     &MBConv6_12_EXPD_BN_VARIANCE,
                &MBConv6_12_EXPD_BN_WEIGHTS,  &MBConv6_12_EXPD_BN_BIAS,
                &MBConv6_12_DW_BN_MEAN,       &MBConv6_12_DW_BN_VARIANCE,
                &MBConv6_12_DW_BN_WEIGHTS,    &MBConv6_12_DW_BN_BIAS,
                &MBConv6_12_PRJ_BN_MEAN,      &MBConv6_12_PRJ_BN_VARIANCE,
                &MBConv6_12_PRJ_BN_WEIGHTS,   &MBConv6_12_PRJ_BN_BIAS);   
  


  // MBConv6_13 layer implementation

  Matrix ConvOut13;
  MBConv_Layer(&ConvOut12, &ConvOut13,
                &D_MBConv_6_13_EXPD_WEIGHTS,  &D_MBConv_6_13_DW_WEIGHTS,
                &D_MBConv_6_13_SQZ_1_WEIGHTS, &D_MBConv_6_13_SQZ_2_WEIGHTS,
                &D_MBConv_6_13_PRJ_WEIGHTS,
                MBCONV_6_13_EXPD_F_DENSITY,   MBCONV_6_13_DW_F_DENSITY, 
                MBCONV_6_13_SQZ_1_F_DENSITY,  MBCONV_6_13_SQZ_2_F_DENSITY, 
                MBCONV_6_13_PRJ_F_DENSITY,
                ConvOut12.depth,              MBCONV_6_13_PRJ_F_DENSITY, MBCONV_6_13_DW_F_HEIGHT,
                MBCONV_6_13_STRIDE,           MBCONV_6_13_PADDING, MBCONV_6_13_SKIP,
                &MBConv6_13_SQZ_1_bias,       &MBConv6_13_SQZ_2_bias,
                &MBConv6_13_EXPD_BN_MEAN,     &MBConv6_13_EXPD_BN_VARIANCE,
                &MBConv6_13_EXPD_BN_WEIGHTS,  &MBConv6_13_EXPD_BN_BIAS,
                &MBConv6_13_DW_BN_MEAN,       &MBConv6_13_DW_BN_VARIANCE,
                &MBConv6_13_DW_BN_WEIGHTS,    &MBConv6_13_DW_BN_BIAS,
                &MBConv6_13_PRJ_BN_MEAN,      &MBConv6_13_PRJ_BN_VARIANCE,
                &MBConv6_13_PRJ_BN_WEIGHTS,   &MBConv6_13_PRJ_BN_BIAS);


  Matrix ConvOut14;
  MBConv_Layer(&ConvOut13, &ConvOut14,
                &D_MBConv_6_14_EXPD_WEIGHTS,  &D_MBConv_6_14_DW_WEIGHTS,
                &D_MBConv_6_14_SQZ_1_WEIGHTS, &D_MBConv_6_14_SQZ_2_WEIGHTS,
                &D_MBConv_6_14_PRJ_WEIGHTS,
                MBCONV_6_14_EXPD_F_DENSITY,   MBCONV_6_14_DW_F_DENSITY, 
                MBCONV_6_14_SQZ_1_F_DENSITY,  MBCONV_6_14_SQZ_2_F_DENSITY, 
                MBCONV_6_14_PRJ_F_DENSITY,
                ConvOut13.depth,              MBCONV_6_14_PRJ_F_DENSITY, MBCONV_6_14_DW_F_HEIGHT,
                MBCONV_6_14_STRIDE,           MBCONV_6_14_PADDING, MBCONV_6_14_SKIP,
                &MBConv6_14_SQZ_1_bias, 	    &MBConv6_14_SQZ_2_bias,
                &MBConv6_14_EXPD_BN_MEAN,     &MBConv6_14_EXPD_BN_VARIANCE,
                &MBConv6_14_EXPD_BN_WEIGHTS,  &MBConv6_14_EXPD_BN_BIAS,
                &MBConv6_14_DW_BN_MEAN,       &MBConv6_14_DW_BN_VARIANCE,
                &MBConv6_14_DW_BN_WEIGHTS,    &MBConv6_14_DW_BN_BIAS,
                &MBConv6_14_PRJ_BN_MEAN,      &MBConv6_14_PRJ_BN_VARIANCE,
                &MBConv6_14_PRJ_BN_WEIGHTS,   &MBConv6_14_PRJ_BN_BIAS);  


  Matrix ConvOut15;
  MBConv_Layer(&ConvOut14, &ConvOut15,
                &D_MBConv_6_15_EXPD_WEIGHTS,  &D_MBConv_6_15_DW_WEIGHTS,
                &D_MBConv_6_15_SQZ_1_WEIGHTS, &D_MBConv_6_15_SQZ_2_WEIGHTS,
                &D_MBConv_6_15_PRJ_WEIGHTS,
                MBCONV_6_15_EXPD_F_DENSITY,   MBCONV_6_15_DW_F_DENSITY, 
                MBCONV_6_15_SQZ_1_F_DENSITY,  MBCONV_6_15_SQZ_2_F_DENSITY, 
                MBCONV_6_15_PRJ_F_DENSITY,
                ConvOut14.depth,              MBCONV_6_15_PRJ_F_DENSITY, MBCONV_6_15_DW_F_HEIGHT,
                MBCONV_6_15_STRIDE,           MBCONV_6_15_PADDING, MBCONV_6_15_SKIP,
                &MBConv6_15_SQZ_1_bias,       &MBConv6_15_SQZ_2_bias,
                &MBConv6_15_EXPD_BN_MEAN,     &MBConv6_15_EXPD_BN_VARIANCE,
                &MBConv6_15_EXPD_BN_WEIGHTS,  &MBConv6_15_EXPD_BN_BIAS,
                &MBConv6_15_DW_BN_MEAN,       &MBConv6_15_DW_BN_VARIANCE,
                &MBConv6_15_DW_BN_WEIGHTS,    &MBConv6_15_DW_BN_BIAS,
                &MBConv6_15_PRJ_BN_MEAN,      &MBConv6_15_PRJ_BN_VARIANCE,
                &MBConv6_15_PRJ_BN_WEIGHTS,   &MBConv6_15_PRJ_BN_BIAS);   

  // Head layer
  Matrix HEAD_OUT;
  HEAD_LAYER(&ConvOut15, &HEAD_CONV_WEIGHTS, &HEAD_FC_WEIGHTS,
              HEAD_CONV_F_HEIGHT, HEAD_CONV_F_WIDTH, HEAD_CONV_F_DEPTH, HEAD_CONV_F_DENSITY,
              0, 1,
              &HEAD_OUT);
  
}



// The last layer in efficient net
void HEAD_LAYER(Matrix *INPUT_MATRIX, Matrix *F_HEAD, Matrix *FC_WEIGHTS,
                int filter_height, int filter_width, int filter_depth, int filter_density,
                int padding, int stride,
                Matrix *HEAD_OUT)
{                
  // Calculate output dimensions       
  int out_height = (INPUT_MATRIX -> height + 2 * padding - filter_height) / stride + 1;
  int out_width = (INPUT_MATRIX -> width + 2 * padding - filter_width) / stride + 1;
  int out_depth = filter_density;

  Set_DeviceMatrix(out_height, out_width, out_depth, HEAD_OUT,
                   "Output is allocated in device memory"); 

  // 1st 3 layers: Conv2d 1x1: BN: Swish()
  Conv2d_Layer(INPUT_MATRIX,  F_HEAD, HEAD_OUT,
              stride, padding,
              INPUT_MATRIX -> depth, out_depth, filter_density,
              Conv2d_1_x_1, NO_ACTIVATION,
              0, NULL);
 
  BN_ALL_PRE_DEFINED(HEAD_OUT, SWISH_ACTIVATION, 
                      &D_HEAD_BN_MEAN,	&D_HEAD_BN_VARIANCE ,
                      &D_HEAD_BN_WEIGHTS, &D_HEAD_BN_BIAS);


  // 4th layer: Average pooling layer which is just a reduction sum layer
  // Get mean values for all channels; Dims(1 x 1 x InputDepth)
  
  Matrix MEAN, Result_Mean;

  Set_DeviceMatrix(HEAD_OUT -> depth,
                    (int)ceil((double)HEAD_OUT -> height * HEAD_OUT -> width / (2 * BLOCK_SIZE)),
                    1, 
                    &Result_Mean, 
                    "Reesult Mean matrix allocated in device memory");

  REDUCTION_SUM(HEAD_OUT, &MEAN, &Result_Mean);


  // 5th layer: Fully connected layer::

  // Set Output matrix details
  Matrix Out1;
  Set_DeviceMatrix(1, 1000, 1, &Out1, "Setting Final Model Output matrix in device memory");
     
  Conv_vidMultiplier(&Out1, FC_WEIGHTS, &Result_Mean,
                      1, 1000, 1,
                      Conv2d_1_x_1, 1,
                      NO_ACTIVATION, 
                      0, NULL);
  
  stop("Model: ", 0);
  
  Matrix tmp_out_host;
  set_allocate_Host(&tmp_out_host, 1, 1000, 1);
  just_copy_DTH(&tmp_out_host, &Out1, "Copying to add bias");
 
  for (int i = 0; i < 1000; i++)
  {
    tmp_out_host.elements[i] += Head_linear_bias[i];
  }

  just_copy_HTD(&Out1, &tmp_out_host, "Copying to add bias");
  show_me_enhanced_from_devince(&Out1, "Model final output::");
}

// The first layer in efficient net: 
// It reutnrs a pointer to matrix, its elements are allocated in device memory 
void STEM_LAYER(Matrix *DInput_Mat, Matrix *F_STEM,
                  int image_height, int image_width, int image_depth,
                  int filter_height, int filter_width, int filter_depth, int filter_density,
                  int padding, int stride,
                  Matrix *STEM_OUT)
{

  // Calculate output dimensions       
  int out_height = (image_height + 2 * padding - filter_height) / stride + 1;
  int out_width = (image_width + 2 * padding - filter_width) / stride + 1;
  int out_depth = filter_density;
 

  // Allow the output from this layer to go accross the next layer       
  Set_DeviceMatrix(out_height, out_width, out_depth, STEM_OUT,
                   "Output is allocated in device memory"); 
 
  Conv2d_Layer(DInput_Mat,  F_STEM, STEM_OUT,
              stride, padding,
              image_depth, out_depth, filter_density,
              Regular_Conv, NO_ACTIVATION,
              0, NULL);
 

  BN_ALL_PRE_DEFINED(STEM_OUT, SWISH_ACTIVATION, 
                      &D_STEM_BN_MEAN, &D_STEM_BN_VARIANCE ,
                      &D_STEM_BN_WEIGHTS, &D_STEM_BN_BIAS);  
}