#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <hipsolver.h>
#include <hip/hip_runtime.h>

#include "/content/MBCONVS_float/functionsV2.h"
#include "/content/MBCONVS_float/KERNELSH.h"

/* Kernel definitions */
__global__ void INPUT_UNROLLING(int stride, int Filter_Height,
                                float *Input, int H1, int W1, int D1,
                                float *X_unrolled, int H2, int W2, int D2,
                                int Output_Height, int Output_Width)
{  
    int bx = blockIdx.x, by = blockIdx.y, bz = blockIdx.z;
    int tx = threadIdx.x, ty = threadIdx.y;
 
    // Select row and column values 
    int row =  by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    int depth = bz;
 
    int col_no_strided = col, row_no_strided = row;
    int depth_offset = depth * W2 * Filter_Height * Filter_Height;

    /* 
      Note for bx, by and bz= 0, stride = 2: 
          @ tx = 0, ty = 0 -> First multiply the col * stride, row * stride; = 0, 0
                            you are shifting in x direction using local col
                            you are shifting in y direction using local row;
          @ tx = 1, ty = 0 -> First multiply the col * stride, row * stride; = 2, 0 
                            you are shifting in x direction using local col
                            you are shifting in y direction using local row;   
          @ tx = 0, ty = 1 -> First multiply the col * stride, row * stride; = 0, 2 
                            you are shifting in x direction using local col
                            you are shifting in y direction using local row;                   
    */ 
  
    col *= stride; row *= stride;
 
    // Limit number of threads 
    if (row_no_strided < Output_Height && col_no_strided < Output_Width && depth < D1)
    {   
      // Each thread unrolls k x k elements
      for (int local_row = 0; local_row < Filter_Height; local_row++)
      {
        for (int local_col = 0; local_col < Filter_Height; local_col++)
        {                                  
          // 1. local row and column shifts affect the locations in Unrolled matrix
          // 2. For each col and row non strided values -> you are adding an offset to columns and rows in Unrolled matrix
          // 3. Offset the depth using "depth_offset" variable
          X_unrolled[local_col * W2 + local_row * Filter_Height * W2 + col_no_strided + row_no_strided * Output_Width + depth_offset] = 
          Input[(row + local_row) * W1 + (col + local_col) + depth * H1 * W1];
        }
      }
    }

}

__global__ void DWConv2d_kernel(float *Input, int H1, int W1, int D1,
                                float *Filter, int H2, int W2, int D2,
                                float *Output, int H3, int W3, int D3,
                                int stride)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    int dep = bz;

    float Pvalue = 0;

    if (row < H3 && col < W3 && dep < D3)
    {
      // 1 thread unrolls kxk section
      for (int j = 0; j < H2; j++)
      {
        for (int i = 0; i < W2; i++)
        {
            Pvalue += Filter[j * W2 + i + dep * H2 * W2] *
                Input[(j * W1 + row * stride * W1) + (i + col * stride) + dep * H1 * W1];
        }
      }
      Output[row * W3 + col + dep * H3 * W3] = Pvalue;
    }

}

__global__ void MatrixMulKernel(float *M, int H1, int W1, int D1,
                                float *N, int H2, int W2, int D2,
                                float *P, int H3, int W3, int D3,
                                int num_blocks, int activation, 
                                int IS_BIASED, float *bias_mat)
{
  __shared__ float Mds[Tile_GEMM][Tile_GEMM];
  __shared__ float Nds[Tile_GEMM][THREAD_GRANULARITY_BLOCKS * Tile_GEMM];

  int bx = blockIdx.x * THREAD_GRANULARITY_BLOCKS;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Identify the row and column of the d_P element to work on
  int Row = by * Tile_GEMM + ty;
  int Col = bx * Tile_GEMM + tx;
  float Pvalue = 0;
  float Pvalue_2 = 0;

  // Loop over the d_M and d_N tiles required to compute d_P element
  for (int ph = 0; ph < num_blocks; ++ph)
  {
    // Collaborative loading of d_M and d_N tiles into shared memory
    if ((Row < H1) && (ph * Tile_GEMM + tx) < W1)
    {
      Mds[ty][tx] = M[Row * W1 + ph * Tile_GEMM + tx];
    }

    if ((ph * Tile_GEMM + ty) < H2 && Col < W2)
    {
      Nds[ty][tx] = N[(ph * Tile_GEMM + ty) * W2 + Col];
    }

    if ((ph * Tile_GEMM + ty) < H2 && Col + Tile_GEMM < W2)
    {
      Nds[ty][tx + Tile_GEMM] = N[(ph * Tile_GEMM + ty) * W2 + Col + Tile_GEMM];
    }     
   
    __syncthreads();

    for (int k = 0; k < Tile_GEMM && (ph * Tile_GEMM) + k < W1; ++k)
    {
      Pvalue += Mds[ty][k] * Nds[k][tx];
      if (Col + Tile_GEMM < W2)
        Pvalue_2 += Mds[ty][k] * Nds[k][tx + Tile_GEMM];
    }
  
    __syncthreads();

  }

  if ((Row < H1) && (Col < W2))
  {
    P[Row * W3 + Col] = Pvalue;
    
    switch (IS_BIASED) 
    {
      case BIASED:
        Pvalue = Pvalue + bias_mat[Row];
        break;
      
      default:
        break;
    } 
            
    switch (activation) 
    {
      case SWISH_ACTIVATION:
        // Swish activation function
        P[Row * W3 + Col] = Pvalue / (1.0f + expf(-1.0f * Pvalue));
        break;

      case SIGMOID_ACTIVATION:
        // Sigmoid activation function
        P[Row * W3 + Col] = 1.0f / (1.0f + expf(-1.0f * Pvalue));
        break;

      default:
        break;
    }
  }

  if ((Row < H1) && (Col + Tile_GEMM < W2))
    {
      P[Row * W3 + Col + Tile_GEMM] = Pvalue_2;
      
      switch (IS_BIASED) 
      {
        case BIASED:
          Pvalue_2 = Pvalue_2 + bias_mat[Row];
          break;
        
        default:
          break;
      } 
              
      switch (activation) 
      {
        case SWISH_ACTIVATION:
          // Swish activation function
          P[Row * W3 + Col + Tile_GEMM] = Pvalue_2 / (1.0f + expf(-1.0f * Pvalue_2));
          break;

        case SIGMOID_ACTIVATION:
          // Sigmoid activation function
          P[Row * W3 + Col + Tile_GEMM] = 1.0f / (1.0f + expf(-1.0f * Pvalue_2));
          break;

        default:
          break;
      }
    }    
    
}


__global__ void ConvChannelElementWiseMultiplication(float *A, int H1, int W1, int D1,
                                                     float *B)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int depth = blockIdx.z;

    int index = depth * W1 * H1 + row * W1 + col;

    if ((row < H1) && (col < W1) && (depth < D1))
    {
        A[index] = A[index] * B[depth];
    }
}

__global__ void CastingDivision(float *A, int W1, float B)
{
    // Warning: 1-D kernel only in x dir.
 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
 
    if ((col < W1))
    {
        A[col] /= B;
    }
}

// Used with MBConv layers that has skip identity = true
__global__ void Identity_Skip(float *A,  int H1, int W1, int D1,
                              float *B)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int depth = blockIdx.z;

    int index = depth * W1 * H1 + row * W1 + col;

    if ((row < H1) && (col < W1) && (depth < D1))
    {
        A[index] = A[index] + B[index];
    }
}

__global__ void Complete_Padding_Process(float *Original_Padded, int H1, int W1, int D1, 
                                         float *Original,        int H2, int W2, int D2,
                                         int padding_value)
{   
    // There must be a constant shift between indeces in 2 matrices
    // The code is based on x axis only
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int depth = blockIdx.z * blockDim.z + threadIdx.z;

    int index = depth * W2 * H2 + row * W2 + col;
    int Padding_Index = depth * W1 * H1 + (row + padding_value) * W1 + (col + padding_value);

    if ((row < (H2)) && (col < (W2)) && (depth < (D2)))
    {
        Original_Padded[Padding_Index] = Original[index];
    }
}

/* Batch Normalization Kernels */
const int BLOCK_SIZE = 128;

__global__ void BN_Kernel_Mean_Reduction(float *input, int H1, int W1, int D1,
                                         float *Mean, int W2)
{
    /*
        This code works on 2 * Block_Size elements.
        i.e. for 512 Block_Size -> we are reducing 1024 elements.
        Each thread loads 2 elements, one at tx and the
        other shifted by blockIdx.x.
    */

    __shared__ float partialSum[2 * BLOCK_SIZE];
    float tmp = 0;

    int tx = threadIdx.x;
    int bx = blockDim.x;

    int by_index = blockIdx.y;
    int bx_index = blockIdx.x;

    // The start variable is to get offset for input matrix in loading
    int start = blockIdx.x * (2 * blockDim.x);
    int start_yDir = blockIdx.y * W1;

    if (start + tx < W1 && start_yDir < H1 * W1)
        // Load 2 elements in the shared memory
        partialSum[tx] = input[start + tx + start_yDir];
    else
        partialSum[tx] = tmp;

    if (tx + bx + start < W1 && start_yDir < H1 * W1)
        partialSum[bx + tx] = input[start + bx + tx + start_yDir];
    else
        partialSum[bx + tx] = tmp;


    unsigned int stride = 0;

    __syncthreads();

    for (stride = blockDim.x; stride > 0; stride = stride / 2.0f)
    {
        __syncthreads();
        if (tx < stride)
            partialSum[tx] += partialSum[tx + stride];
    }

    __syncthreads();


    if (tx == 0)
        Mean[bx_index + by_index * W2] = partialSum[tx];

}

__global__ void ElementWiseSquaring(float *A, int H1, int W1, int D1)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int depth = blockIdx.z;

    int index = depth * W1 * H1 + row * W1 + col;

    if ((row < H1) && (col < W1) && (depth < D1))
    {
        A[index] = A[index] * A[index];
    }
}

__global__ void ElementWiseSubtraction(float *A, int H1, int W1, int D1,
                                       float *mean)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x  + threadIdx.x;
    int depth = blockIdx.z;

    int index = depth * W1 * H1 + row * W1 + col;

    if ((row < H1) && (col < W1) && (depth < D1))
    {
        A[index] = A[index] - mean[depth];
    }
}


__global__ void BN_Kernel_Final_Layer(float *A, int H1, int W1, int D1, 
                                      float *D_mean, float *D_variance,
                                      float *D_weight, float *D_bias,
                                      int activate)
{
    // Activate values are assigned as follow
    /*
      0 -> no activation, 1 -> swish, 2 -> sigmoid
    */
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int depth = blockIdx.z;

    int index = depth * W1 * H1 + row * W1 + col;
    int index3 = depth;

    float tmp = 0;
 
    if ((row < H1) && (col < W1) && (depth < D1))
    {
        A[index] = ((A[index] - D_mean[index3]) / (sqrtf(D_variance[index3] + 0.001f))) * D_weight[index3] + D_bias[index3];
        tmp = A[index];

        switch (activate) {
                  case 1:
                      // Swish activation function
                      A[index] = tmp / (1.0f + expf(-1.0f * tmp));
                      break;
                  case 2:
                      // Sigmoid activation function
                      A[index] = 1.0f / (1.0f + expf(-1.0f * tmp));
                      break;
                  default:
                      break;
                    }
    }
}