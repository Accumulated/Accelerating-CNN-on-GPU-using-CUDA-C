#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <hipsolver.h>
#include <hip/hip_runtime.h>


#include "/content/MBCONVS_float/functionsV2.h"
#include "/content/MBCONVS_float/KERNELSH.h"

static void HandleError( hipError_t err,
                         char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

float time_defined = 0, tmp_time = 0, total_time_for_layer = 0;; 
hipEvent_t start_timing, stop_timing;


int show_out = 0;

int total_constant_memory = 0;
                
// Device memory for filters
void DEFINE_FILTERS_FOR_MBCONV(Matrix *D_f1, float *filter1, int h1, int w1, int dens1,
                               Matrix *D_f2, float *filter2, int h2, int w2, int dens2,
                               Matrix *D_f3, float *filter3, int h3, int w3, int dens3,
                               Matrix *D_f4, float *filter4, int h4, int w4, int dens4,
                               Matrix *D_f5, float *filter5, int h5, int w5, int dens5)
{
    // Note: No allocations are done, just pointers point to matrices pre-defined

    // This condition is important as the float * is NULL
    if (MBCONV1_0_flag == 1);
    else
      set_allocate_copy_array_Device(D_f1, filter1,
                                    h1, w1, dens1,
                                    "1st filter allocated");
 
    set_allocate_copy_array_Device(D_f2, filter2,
                                    h2, w2, dens2,
                                    "2nd filter allocated");
 
    set_allocate_copy_array_Device(D_f3, filter3,
                                    h3, w3, dens3,
                                    "3rd filter allocated");

    set_allocate_copy_array_Device(D_f4, filter4,
                                    h4, w4, dens4,
                                    "4th filter allocated");

    set_allocate_copy_array_Device(D_f5, filter5,
                                    h5, w5, dens5,
                                    "5th filter allocated");                                                         
}

// Free the device filters
void FREE_FILTERS_FOR_MBCONV(Matrix *D_f1, Matrix *D_f2, 
                             Matrix *D_f3, Matrix *D_f4,
                             Matrix *D_f5)
{
  hipFree(D_f1 -> elements);
  hipFree(D_f2 -> elements);
  hipFree(D_f3 -> elements);
  hipFree(D_f4 -> elements);
  hipFree(D_f5 -> elements);
}

void REDUCTION_SUM(Matrix* Output_Modified, Matrix *sum, Matrix *DMean)
{
    /*
      The mean will be a row vector of 1 x C;
      where C is number of original matrix channels
      All input matrices for this function are device matrices,
      except for sum, it's just a transition that later can be removed 
    */
    
    // Define number of blocks in different directions
    int nbx = 0;
    int nby = 0;
    int nbz = 1;

    size_t size;
    hipError_t err;
 
    /*
      Load input Matrix inot device t calculate mean for it
      Unfortionately the code requires to copy the input matrix
    */
 
    Matrix DInputMat;
    // Allocate and set its dimensions as needed from the algorithm
    Set_DeviceMatrix(Output_Modified -> depth, Output_Modified -> height * Output_Modified -> width, 1,
                     &DInputMat, "Copyting Input of reduction mean function into Device memory");

    // Copy input from a device memory to device memory in order to process the elements
    size = DInputMat.height * DInputMat.width * DInputMat.depth * sizeof(float);
    err = hipMemcpyAsync(DInputMat.elements, Output_Modified -> elements, size, hipMemcpyDeviceToDevice, 0);
    CheckCudaError("Copying mean matrix elements from ", err);

    /* Starting reduction sum calculations */

    // Note: All blocks are 1D threads. We use Block.x to reduce 1 channel elements'
    // Diffferent block.y to address different number of channels
    nbx = (int)ceil((float)DInputMat.width / (2 * BLOCK_SIZE));
    nby = (int)ceil((float)DInputMat.height);

    if (nbx == 0) nbx = 1;
    if (nby == 0) nby = 1;

    // For loop is held to maintain huge number of summations needed
    for (int i = 0; DInputMat.width != 1; i++)
    {
        dim3 dim_Grid2(nbx, nby, nbz);
        dim3 dim_Block2(BLOCK_SIZE, 1, 1);

        // Make sure to synch between multiple runs
       //hipDeviceSynchronize();
        
        BN_Kernel_Mean_Reduction <<< dim_Grid2, dim_Block2 >>> (DInputMat.elements,
                                                                DInputMat.height,
                                                                DInputMat.width,
                                                                DInputMat.depth,
                                                                DMean -> elements,
                                                                DMean -> width);

        
        // Save and copy mean values array into the filter array
        size = DMean -> height * DMean -> width * DMean -> depth * sizeof(float);
        err = hipMemcpyAsync(DInputMat.elements, DMean -> elements, size, hipMemcpyDeviceToDevice, 0);
        CheckCudaError("Copying mean matrix elements from ", err);

        // Modify filter width to fit into the new elements width
        DInputMat.width = nbx;
        DInputMat.height = nby;

        // Recalculate number of blocks in x direction
        nbx = (int)ceil((float)DInputMat.width / (2 * BLOCK_SIZE));
        nby = (int)ceil((float)DInputMat.height);

        if (nbx == 0) nbx = 1;
        if (nby == 0) nby = 1;

        // Set width of mean matrix to the new number of blocks
        DMean -> width = nbx;
        DMean -> height = nby;
    }
    
    // Set mean matrix to 1 X C X 1 to ease further calculations
    DMean -> height = 1; DMean -> width = Output_Modified -> depth; DMean -> depth = 1;
    
    nbx = (int)ceil((float)DMean -> width / 1024);
    
    dim3 dim_Grid2(nbx, 1, 1);
    dim3 dim_Block2(1024, 1, 1);
    CastingDivision <<<dim_Grid2, dim_Block2>>> (DMean -> elements, DMean -> width, 
                                                 Output_Modified->height * Output_Modified->width);
}


/*  Squeeze_and_Excite(&tmp2, &SE_OUT, F3, F4,
                      FD4, FD3, FD4, FD3);
                      */
// Note: input and output channels args represents the 2 Conv layers output channels respectively
void Squeeze_and_Excite(Matrix* InputIMG, Matrix* Result,
                        Matrix* Filter1, Matrix* Filter2,
                        int FilterDensity2, int FilterDensity1,
                        int input_channels, int output_channels,
                        Matrix * First_bias, Matrix *Second_bias)
{
    /*
       Steps in squeeze and excite layer:
        1. Get mean value for a tensor
        2. pass the mean to the covolution, swish, convolution, sigmoid
        3. the result will be a 1 x 1 x C, multiply elementwise.
          "each element in a channel is multiplied by the result's corresponding channel element"
        
        Filter Density means #filters used
 
      Note: All input matrices are device allocated matrices
    */

 
    /*
      Get mean values for all channels; Dims(1 x InputDepth x 1) 
      Note: Mean matrix is a host allocated memory in REDUCTION_SUM;
            It's used to get the final summation from device and
            then divide each element sequentially by total number 
            of elements. It's then later copied back to Result_Mean
            Matrix which is a device matrix.
            "This can be later changed"
    */
 
    Matrix MEAN, Result_Mean;

    Set_DeviceMatrix(InputIMG -> depth,
                      (int)ceil((double)InputIMG -> height * InputIMG -> width / (2 * BLOCK_SIZE)),
                      1, 
                      &Result_Mean, 
                      "Reesult Mean matrix allocated in device memory");

    REDUCTION_SUM(InputIMG, &MEAN, &Result_Mean);
 

    // Tmp1 is used as a transition between 2 convolution layers; Dims(1 x 1 x FilterDensity3)
    Matrix tmp1;
    Set_DeviceMatrix(1, 1, FilterDensity1, &tmp1, "Allocating tmp1 in device for transition");
 
    // tmp2 matrix is the result from sigmoid function: Dims(1 x 1 x FilterDensity4)
    Matrix tmp2;
    Set_DeviceMatrix( 1, 1, FilterDensity2, &tmp2, "Allocating tmp2 in device for final output");
 
    // Sequence: Conv1x1, swish, Conv1x1, sigmoid 
    // Warning: Remember to pre-process Result_Mean matrix to match 1 x 1 x C as it's the input in this case to Conv2d
    Set_HostMatrix(1, 1, InputIMG -> depth, &Result_Mean);
 
    Conv2d_Layer(&Result_Mean, Filter1, &tmp1, 1, 0, input_channels, output_channels, FilterDensity1,
                 Conv2d_1_x_1, SWISH_ACTIVATION,
                 BIASED, First_bias);
    
    Conv2d_Layer(&tmp1, Filter2, &tmp2, 1, 0, output_channels, input_channels, FilterDensity2,
                 Conv2d_1_x_1, SIGMOID_ACTIVATION,
                 BIASED, Second_bias);
 

    int nbx = (int)ceil((float)InputIMG -> width / DYNAMIC_TILE);
    int nby = (int)ceil((float)InputIMG -> height / DYNAMIC_TILE);
    int nbz = InputIMG -> depth;

    if (nbx == 0) nbx = 1;

    if (nby == 0) nby = 1;

    // This is the only kernel that runs 3d Grid; 
    // Each block in z dimension controls 1 channel  
    dim3 dim_Grid2(nbx, nby, nbz);
    dim3 dim_Block2(DYNAMIC_TILE, DYNAMIC_TILE, 1);

    // C then D, the final multiplication is in C matrix
    ConvChannelElementWiseMultiplication <<< dim_Grid2, dim_Block2 >>> (InputIMG -> elements,
                                                                        InputIMG -> height,
                                                                        InputIMG -> width,
                                                                        InputIMG -> depth,
                                                                        tmp2.elements);

   
    hipFree(tmp1.elements);
    hipFree(tmp2.elements);
}

// Warning: Fuction Input matrices are allocated in device memory directly
// InputIMG, FilterK and ConvOut are device memory allocations
void Conv2d_Layer(Matrix* InputIMG, Matrix* FilterK, Matrix* ConvOut,
                  int stride, int padding,
                  int InputChannels, int OutputChannels, int FilterDensity,
                  int Conv_Type, int activation_type,
                  int BIASED_CHOISE, Matrix *biasMat)
{
    //printf("The start of Conv2d layer\n\n");
    
    int OutputHeight = 0, OutputWidth = 0, OutputDepth = 0;

    // 1x1 Conv2d is a special case of Convolution
    if (Conv_Type == Conv2d_1_x_1)
    {
        // Conv2d 1x1 has stride = 1, no padding and K = 1

        /*
          Input Dimensions is the same as Output dimensions
          Only Depth of the output channels differ from input
        */
        OutputHeight = InputIMG -> height; OutputWidth = InputIMG -> width; OutputDepth = FilterDensity;

        /*
          Note: Set_HostMatrix function just changes the dimensions
                so it's okey to use on a device memory
        */
     
        // Modify Filter Matrix to have dimensions ((K^2 * M) x C x 1); K = 1
        Set_HostMatrix(1 * 1 * FilterDensity, InputIMG -> depth, 1, FilterK);

        // Modify Input matrix to have dimensions (C x (H * W) x 1)
        Set_HostMatrix(InputIMG -> depth, InputIMG -> height * InputIMG -> width, 1, InputIMG);

        // Modify Output Matrix preprocessing to have dimesions ((K^2 * M) x (H * W) x 1); K = 1
        Set_HostMatrix(1 * 1 * FilterDensity, OutputWidth * OutputHeight, 1, ConvOut);

        Conv_vidMultiplier(ConvOut, InputIMG, FilterK,
                            OutputHeight, OutputWidth, OutputDepth,
                            Conv2d_1_x_1, 1,
                            activation_type, 
                            BIASED_CHOISE, biasMat);
    }
    else if (Conv_Type == DWConv_k_x_k)
    {
        // Ptr is used to alternate between input image and padding if needed
        Matrix* ptr = InputIMG;

        // DWConv2d has stride = s, padding = p and kernel = k
        OutputHeight = ConvOut -> height; OutputWidth = ConvOut -> width; OutputDepth = ConvOut -> depth;

        Matrix padded_matr;
        if (padding != 0)
        {
            Padding_Zeros_Function(InputIMG, padding, &padded_matr);
            ptr = &padded_matr;
        }
       
        Conv_vidMultiplier(ConvOut, ptr, FilterK,
                            OutputHeight, OutputWidth, OutputDepth,
                            DWConv_k_x_k, stride,
                            activation_type, 
                            BIASED_CHOISE, biasMat);

        // Padded matrix is no longer needed as Convout has the final result
    }
    // Any other kernel size goes here
    else
    {        
        // Regular convolution: Filter and input unrolling
        Matrix* ptr = InputIMG;
        OutputHeight = (ptr -> height + 2 * padding - FilterK -> height) / stride + 1;
        OutputWidth = (ptr -> width + 2 * padding - FilterK -> width) / stride + 1;
        OutputDepth = FilterDensity;

        Matrix padded_matr;
        if (padding != 0)
        {
            Padding_Zeros_Function(InputIMG, padding, &padded_matr);
            ptr = &padded_matr;          
        }

        // 1st phase: Filter unrolling

        // Unrolled filter has dimesnios (M x (C * k * k) x 1)
        Set_HostMatrix(FilterDensity, FilterK -> depth / FilterDensity * FilterK -> height * FilterK -> width,
                      1, FilterK);

        // 2nd phase: Input unrolling

        // The unrolled Input matrix has dimensions((C * k * k) x (H_out * W_out) x 1)
        Matrix INPUT_MODIFIED;
        
        Set_DeviceMatrix(ptr -> depth * 3 * 3,
                        OutputHeight * OutputWidth, 1,
                        &INPUT_MODIFIED, 
                        "Input unrolled Matrix allocated in device memory");

        Input_Unroll_gpu(stride, ptr, &INPUT_MODIFIED, OutputHeight, OutputWidth, 3);

        // Convolution output has dimensions of (M x (H_out * W_out) x 1)
        Set_HostMatrix(FilterDensity, OutputWidth * OutputHeight, 1, ConvOut);

          
        // Perform Multiplication and re-edit the dimensions of output
        Conv_vidMultiplier(ConvOut, &INPUT_MODIFIED, FilterK,
                            OutputHeight, OutputWidth, OutputDepth,
                            Regular_Conv, stride,
                            activation_type,
                            BIASED_CHOISE, biasMat);

    }
 }

// 5 Filters needed to run the 4 layers sequentially
void MBConv_Layer(Matrix* Input, Matrix* MBConvOut,
    Matrix* F1, Matrix* F2, Matrix* F3, Matrix* F4, Matrix* F5,
    int FD1, int FD2, int FD3, int FD4, int FD5,
    int input_channels, int output_channels, int FilterSizeDW,
    int Stride, int padding, int skip,
    Matrix *bias1, Matrix *bias2,
    Matrix *MBConv_expansion_conv_BN_mean,     Matrix *MBConv_expansion_conv_BN_variance,
    Matrix *MBConv_expansion_conv_BN_weights,  Matrix *MBConv_expansion_conv_BN_bias,
    Matrix *MBConv_depthwise_conv_BN_mean,     Matrix *MBConv_depthwise_conv_BN_variance,
    Matrix *MBConv_depthwise_conv_BN_weights,  Matrix *MBConv_depthwise_conv_BN_bias,
    Matrix *MBConv_project_conv_BN_mean,       Matrix *MBConv_project_conv_BN_variance,
    Matrix *MBConv_project_conv_BN_weights,    Matrix *MBConv_project_conv_BN_bias)
{
    /*
      Note: MBConv1_0 doesn't have the expansion conv function;
            The input matrices to this function are device matrices;
            including all the filters, you don't need to allocate or
            copy any thing; just pass to the functions
    */

    /*
      ptr_mat is the pointer that gets past expansion conv;
      Meaning: in case of MBconv1_0 the pointer is same as input matrix;
                in case of any other MBConv6_! it's the output of Conv2d 
                and BN with swish
    */
    
    Matrix H_OUT;
    Matrix tmp1; Matrix *ptr_mat; 
 
    if (MBCONV1_0_flag == 1)
      ptr_mat = Input;
    else
    {     
      Set_DeviceMatrix(Input -> height, Input -> width , FD1, 
                       &tmp1,
                       "Output_1 is allocated in device memory"); 
               
      // 1st layer: 1x1 Conv2d, stride = 1, padding = 0, K = 1
      Conv2d_Layer(Input, F1, &tmp1, 1, 0,
                   input_channels, FD1, FD1,
                   Conv2d_1_x_1,
                   NO_ACTIVATION, 0, NULL);
  
      BN_ALL_PRE_DEFINED(&tmp1, SWISH_ACTIVATION, 
                          MBConv_expansion_conv_BN_mean,    MBConv_expansion_conv_BN_variance,
                          MBConv_expansion_conv_BN_weights, MBConv_expansion_conv_BN_bias);
      ptr_mat = &tmp1;
    }

   // 2nd Layer: KxK DWconv, stride = s, padding = p, K = k

    // Height and width changes, Only depth remains still
    int OutputHeight = (ptr_mat -> height + 2 * padding - FilterSizeDW)/Stride + 1;
    int OutputWidth = (ptr_mat -> width + 2 * padding - FilterSizeDW)/Stride + 1;
    int OutputDepth = ptr_mat -> depth;
 
    // Set and allocate tmp2 matrix; it's a transistion between expansion and squeeze
    Matrix tmp2;
    Set_DeviceMatrix(OutputHeight, OutputWidth, OutputDepth, &tmp2,
                    "Output_2 is allocated in device memory");    

    Conv2d_Layer(ptr_mat, F2, &tmp2,
                 Stride, padding, FD1, FD2, FD2, DWConv_k_x_k,
                 NO_ACTIVATION, 0, NULL);
  

    BN_ALL_PRE_DEFINED(&tmp2, SWISH_ACTIVATION, 
                       MBConv_depthwise_conv_BN_mean,     MBConv_depthwise_conv_BN_variance,
                       MBConv_depthwise_conv_BN_weights,  MBConv_depthwise_conv_BN_bias);
  
    // 3rd Layer: squeeze and excitation

    /*
      Squeeze excite layer doesn't change the final output dimensions;
      SE_OUT can be removed; Do so later
    */
 
    Matrix *SE_OUT;
    Squeeze_and_Excite(&tmp2, SE_OUT, F3, F4,
                        FD4, FD3, FD4, FD3,
                        bias1, bias2);

    // 4th Layer: 1x1 Conv2d
    // MBConv output pointer is set and finally updated after this layer execution
    Set_DeviceMatrix(tmp2.height, tmp2.width, FD5, MBConvOut,
                     "Matrix final output is allocated in device memory");
 

    // 1x1 Conv2d layer
    Conv2d_Layer(&tmp2, F5, MBConvOut, 1, 0, FD4, FD5, FD5, Conv2d_1_x_1,
                 NO_ACTIVATION, 0, NULL);


    // BatchNorm layer
    BN_ALL_PRE_DEFINED(MBConvOut, NO_ACTIVATION, 
                       MBConv_project_conv_BN_mean,     MBConv_project_conv_BN_variance,
                       MBConv_project_conv_BN_weights,  MBConv_project_conv_BN_bias);

    // Skip identity layer
    if(skip)
    {
      MBConv_SKIP_IDENTITY(MBConvOut, Input);
    }
}


void MBConv_SKIP_IDENTITY(Matrix *parent, Matrix *child)
{
    int nbx = (int)ceil((float)parent -> width / DYNAMIC_TILE);
    int nby = (int)ceil((float)parent -> height / DYNAMIC_TILE);
    int nbz = parent -> depth;

    if (nbx == 0) nbx = 1;

    if (nby == 0) nby = 1;

    // This is the only kernel that runs 3d Grid; 
    // Each block in z dimension controls 1 channel  
    dim3 dim_Grid2(nbx, nby, nbz);
    dim3 dim_Block2(DYNAMIC_TILE, DYNAMIC_TILE, 1);
     
    Identity_Skip <<<dim_Grid2, dim_Block2 >>> (parent -> elements,
                                                  parent -> height,
                                                  parent -> width,
                                                  parent -> depth, 
                                                  child -> elements);
}

void BN_ALL_PRE_DEFINED(Matrix* D_input, int activate, Matrix *mean, Matrix *variance, Matrix *weights, Matrix *bias)
{
    /* The ptr matrix is a device matrix */
     
    /*
      All weights, bias, running mean and running variance
      are pre-defined. Just call the function and use the
      matrices.
      
      All bias, weights, mean and bariance matrices are 1x1xC

      Output Matrix is modified by the equation
      (y = ((x - Mean) / (sqrt(variance) + epsilon)) * weights + bais)
    */

    int nbx = (int)ceil((float)D_input -> width / DYNAMIC_TILE);
    int nby = (int)ceil((float)D_input -> height / DYNAMIC_TILE);
    int nbz = D_input -> depth;

    if (nbx == 0) nbx = 1;
    if (nby == 0) nby = 1;

    // This is the only kernel that runs 3d Grid; 
    // Each block in z dimension controls 1 channel  
    dim3 dim_Grid3(nbx, nby, nbz);
    dim3 dim_Block3(DYNAMIC_TILE, DYNAMIC_TILE, 1);

    BN_Kernel_Final_Layer <<< dim_Grid3, dim_Block3 >>> (D_input -> elements,
                                                         D_input -> height,
                                                         D_input -> width,
                                                         D_input -> depth,
                                                         mean -> elements, variance -> elements,
                                                         weights -> elements, bias -> elements,
                                                         activate);
}

void Padding_Zeros_Function(Matrix* Original_Matrix_Before, int padding_Value, Matrix* padded_Matrix)
{
    /* 
      Note: Matrix coming is a device elemente matrix;
            Original Matrix is a Device input that needs padding
            padded_Matrix is the return of this function;

      Warning: Padded_Matrix has a different size than the Original 
                non padded matrix and it's not allocated in device yet.
                The allocateion is done inside this function.
    */    

    Set_DeviceMatrix(Original_Matrix_Before->height + 2 * padding_Value,
                      Original_Matrix_Before->width + 2 * padding_Value,
                      Original_Matrix_Before->depth,
                      padded_Matrix,
                      "Padded Matrix is allocated in device memory.");

    // 1st: Set padded Matrix with all zeros
    hipMemset(padded_Matrix -> elements,
               0, padded_Matrix->height * padded_Matrix->width * padded_Matrix->depth * sizeof(float)); 

    int nbx = (int)ceil((float)padded_Matrix -> width / DYNAMIC_TILE);
    int nby = (int)ceil((float)padded_Matrix -> height / DYNAMIC_TILE);
    int nbz = padded_Matrix -> depth;

    if (nbx == 0) nbx = 1;

    if (nby == 0) nby = 1;

    dim3 dim_Grid2(nbx, nby, nbz);
    dim3 dim_Block2(DYNAMIC_TILE, DYNAMIC_TILE, 1);

    // Pass to the copying strided kernel to complete the padding process
 
    Complete_Padding_Process <<< dim_Grid2, dim_Block2 >>> (padded_Matrix -> elements,
                                                            padded_Matrix -> height,
                                                            padded_Matrix -> width,
                                                            padded_Matrix -> depth,
                                                            Original_Matrix_Before -> elements,
                                                            Original_Matrix_Before -> height,
                                                            Original_Matrix_Before -> width,
                                                            Original_Matrix_Before -> depth,
                                                            padding_Value);
}


// Call this function directly for 1x1 conv2d. Don't call for DWConv
void Conv_vidMultiplier(Matrix* out_11, Matrix* D_2, Matrix* D_1,
                        int ReconstructOutHieght, int ReconstructOutWidth, int ReconstructOutDepth,
                        int ConvType, int stride_DW, int activation_type, int BIASED_CHOISE, Matrix *biasMat)
{
    /* Note: Out_11, XXX_Trans and Host_Conv_Filter are device matrices */
 
    // The multiplication kernel is used for the 1x1 Conv2d and kxk Conv2d
    if (ConvType == Conv2d_1_x_1 || ConvType == Regular_Conv)
    {    
        // Get number of blocks
        int nbx = (int)ceil((float)out_11 -> width / (THREAD_GRANULARITY_BLOCKS * Tile_GEMM));
        int nby = (int)ceil((float)out_11 -> height / Tile_GEMM);
        int num_block_for_phases = (int)ceil((float)D_1 -> width / Tile_GEMM);

        // Check for zero blocks to make sure code runs correctly
        if (nbx == 0) nbx = 1;
        if (nby == 0) nby = 1;

        dim3 dim_Grid2(nbx, nby, 1);
        dim3 dim_Block2(Tile_GEMM, Tile_GEMM, 1);
     
        if (BIASED_CHOISE == BIASED)
        {
          Set_HostMatrix(out_11 -> height, 1, 1, biasMat);

          // Call shared memory tiled Multiplication  algorithm
          MatrixMulKernel <<< dim_Grid2, dim_Block2 >>> (D_1 -> elements, D_1 -> height, D_1 -> width, D_1 -> depth,
                                                         D_2 -> elements, D_2 -> height, D_2 -> width, D_2 -> depth,
                                                         out_11 -> elements, out_11 -> height, out_11 -> width, out_11 -> depth,
                                                         num_block_for_phases, activation_type,
                                                         BIASED_CHOISE, biasMat -> elements);         
        }
        else
        {
          MatrixMulKernel <<< dim_Grid2, dim_Block2 >>> (D_1 -> elements, D_1 -> height, D_1 -> width, D_1 -> depth,
                                                         D_2 -> elements, D_2 -> height, D_2 -> width, D_2 -> depth,
                                                         out_11 -> elements, out_11 -> height, out_11 -> width, out_11 -> depth,
                                                         num_block_for_phases, activation_type,
                                                         BIASED_CHOISE, NULL);        
         }    
    }

    // This case is for DWConv2d
    else
    {
        int nbx = (int)ceil((float)out_11 -> width / TileDW);
        int nby = (int)ceil((float)out_11 -> height / TileDW);
        int nbz = out_11 -> depth;
     
        if (nbx == 0) nbx = 1;
        if (nby == 0) nby = 1;

        // This is the only kernel that runs 3d Grid; 
        // Each block in z dimension controls 1 channel  
        dim3 dim_Grid2(nbx, nby, nbz);
        dim3 dim_Block2(TileDW, TileDW, 1);


        DWConv2d_kernel << < dim_Grid2, dim_Block2 >> > (D_2 -> elements, D_2 -> height, D_2 -> width, D_2 -> depth,
                                                         D_1 -> elements, D_1 -> height, D_1 -> width, D_1 -> depth,
                                                         out_11 -> elements, out_11 -> height, out_11 -> width, out_11 -> depth,
                                                         stride_DW);          
    }
 
    // Reset the output dimensions to continue in the network
    Set_HostMatrix(ReconstructOutHieght, ReconstructOutWidth, ReconstructOutDepth, out_11);
}

void Input_Unroll_gpu(int st_stride, Matrix* Device_Input, Matrix* Device_Unrolled, int O_H, int O_W, int Filter_Size)
{   
    /* Note: All the function input matrices are device matrices.
            Device_Input matrix is already allocated and ready.
            Device_Unrolled matrix is already allocated and ready. 
    */
    
    int nbx = (int)ceil((float)O_W / TileDW);
    int nby = (int)ceil((float)O_H / TileDW);
    int nbz = Device_Input -> depth;

    if (nbx == 0) nbx = 1;

    if (nby == 0) nby = 1;
 
    dim3 dim_Grid2(nbx, nby, nbz);
    dim3 dim_Block2(TileDW, TileDW, 1);

    // You need to use hipDeviceSynchronize if the kernel isn't working

    INPUT_UNROLLING <<< dim_Grid2, dim_Block2 >>> (st_stride, Filter_Size,
                                                   
                                                   Device_Input -> elements,
                                                   Device_Input -> height,
                                                   Device_Input -> width,
                                                   Device_Input -> depth,

                                                   Device_Unrolled -> elements,
                                                   Device_Unrolled -> height,
                                                   Device_Unrolled -> width,
                                                   Device_Unrolled -> depth,

                                                   O_H, O_W);

    
    //hipDeviceSynchronize(); 

    hipError_t err = hipGetLastError();

    if ( err != hipSuccess )
    {
      printf("CUDA Error: %s\n", hipGetErrorString(err));
      exit(-1);
    } 
}

void DEFINE_FILTERS_FOR_MBCONV_BN(  Matrix *EXP_MEAN, 		  float *filter1, int size_1,
                                    Matrix *EXP_VARIANCE, 	float *filter2, int size_2,
                                    Matrix *EXP_WEIGHTS, 	  float *filter3, int size_3,
                                    Matrix *EXP_BIAS, 		  float *filter4, int size_4,
                                  
                                    Matrix *DW_MEAN, 		    float *filter5, int size_5,
                                    Matrix *DW_VARIANCE, 	  float *filter6, int size_6,
                                    Matrix *DW_WEIGHTS, 		float *filter7, int size_7,
                                    Matrix *DW_BIAS, 		    float *filter8, int size_8,
                                    
                                    Matrix *PRJ_MEAN, 		  float *filter9,  int size_9,
                                    Matrix *PRJ_VARIANCE, 	float *filter10, int size_10,
                                    Matrix *PRJ_WEIGHTS, 	  float *filter11, int size_11,
                                    Matrix *PRJ_BIAS, 		  float *filter12, int size_12)
{
  if (MBCONV1_0_flag);
  else
  {
    set_allocate_copy_array_Device(EXP_MEAN, filter1,
                      size_1, 1, 1,
                      "expand mean"); 
    set_allocate_copy_array_Device(EXP_VARIANCE, filter2,
                      size_2, 1, 1,
                      "expand variance"); 
    set_allocate_copy_array_Device(EXP_WEIGHTS, filter3,
                      size_3, 1, 1,
                      "expand weights"); 
    set_allocate_copy_array_Device(EXP_BIAS, filter4,
                      size_4, 1, 1,
                      "expand bias");
  } 
									  
	set_allocate_copy_array_Device(DW_MEAN, filter5,
									  size_5, 1, 1,
									  "DW mean"); 
	set_allocate_copy_array_Device(DW_VARIANCE, filter6,
									  size_6, 1, 1,
									  "DW variance"); 
	set_allocate_copy_array_Device(DW_WEIGHTS, filter7,
									  size_7, 1, 1,
									  "DW weights"); 
	set_allocate_copy_array_Device(DW_BIAS, filter8,
									  size_8, 1, 1,
									  "expand bias"); 

	set_allocate_copy_array_Device(PRJ_MEAN, filter9,
									  size_9, 1, 1,
									  "DW mean"); 
	set_allocate_copy_array_Device(PRJ_VARIANCE, filter10,
									  size_10, 1, 1,
									  "DW variance"); 
	set_allocate_copy_array_Device(PRJ_WEIGHTS, filter11,
									  size_11, 1, 1,
									  "DW weights"); 
	set_allocate_copy_array_Device(PRJ_BIAS, filter12,
									  size_12, 1, 1,
									  "expand bias"); 
}

// 3 Sequential Operations: Same as "set_allocate_copy_Matrix_Device",
// However, it uses a pointer to float as a parent.
void set_allocate_copy_array_Device(Matrix *child, float *parent,
									int height, int width, int depth,
									char *notification)
{
	Set_DeviceMatrix(height, width, depth, child, notification);

	size_t size = height * width * depth * sizeof(float);
 
	hipError_t err = hipMemcpy(child -> elements, parent, size,
								hipMemcpyHostToDevice);
  
	CheckCudaError(notification, err);
}

// 3 Sequential Operations: Set dimensions, allocate device memory and copy.
void set_allocate_copy_Matrix_Device(Matrix *child, Matrix *parent, char *notification)
{
	Set_DeviceMatrix(parent -> height, parent -> width, parent -> depth,
					          child, notification);

	size_t size = parent -> height * parent -> width * parent -> depth * sizeof(float);
	
	hipError_t err = hipMemcpy(child -> elements, parent -> elements,
								              size, hipMemcpyHostToDevice);
	CheckCudaError(notification, err);
}

void set_allocate_copy_Matrix_Device_specific(Matrix *child, Matrix *parent, char *notification, int height, int width, int depth)
{
	Set_DeviceMatrix(height, width, depth, child, notification);

	size_t size = child -> height * child -> width * child -> depth * sizeof(float);
	

	hipError_t err = hipMemcpy(child -> elements, parent -> elements,
								            size, hipMemcpyHostToDevice);

  
	CheckCudaError(notification, err);
}

void just_copy_HTD(Matrix *child, Matrix *parent, char *notification)
{
    // Read C from device memory
  size_t size = parent -> width * parent -> height * parent -> depth * sizeof(float);
    
	hipError_t err = hipMemcpy(child -> elements, parent -> elements, size, hipMemcpyHostToDevice);

  
	CheckCudaError(notification, err);
}

void just_copy_DTH(Matrix *child, Matrix *parent, char *notification)
{
  // Read C from device memory
  size_t size = parent -> width * parent -> height * parent -> depth * sizeof(float);
  

	hipError_t err = hipMemcpy(child -> elements, parent -> elements, size, hipMemcpyDeviceToHost);
  
	CheckCudaError(notification, err);
}

void set_allocate_Host(Matrix *ptr, int height, int width, int depth)
{
	// Note this function allocates memory, remember to free 
	Set_HostMatrix(height, width, depth, ptr);
	
	int Fsize = height * width * depth* sizeof(float);
 
	ptr -> elements = (float *) malloc(Fsize);
}

void FreeHost_Allocated(Matrix *ptr)
{
	free(ptr -> elements);
}

// Allocations for Device matrices
void Set_DeviceMatrix(int height, int width, int depth, Matrix* ptr, char* NamePtr)
{
    ptr->width = width;
    ptr->height = height;
    ptr->depth = depth;

    size_t size = width * height * depth * sizeof(float);
    hipError_t err = hipMalloc((void **)&(ptr->elements), size);
    CheckCudaError(NamePtr, err);
}

void Set_HostMatrix(int height, int width, int depth, Matrix* ptr)
{
    ptr->width = width;
    ptr->height = height;
    ptr->depth = depth;
}

void CheckCudaError(char* ptr, hipError_t err)
{
    if (err == hipSuccess);
    else
        printf("CUDA error in %s: %s\n", ptr, hipGetErrorString(err));
}


void show_me_enhanced(Matrix* ptr, char* NamePtr)
{
    if(show_out == 1)
    {
      setvbuf(stdout, NULL, _IOLBF, 0);

          printf("%s,"
              "it has height = %d, "
              "width = %d, "
              "depth = %d \n",
              NamePtr, ptr->height, ptr->width, ptr->depth);

          printf("{\n");
          for (int i = 0; i < ptr -> height * ptr -> width * ptr -> depth; i++)
          {
              if (i % ptr->width == 0 && i >= ptr->width)
                  printf("\n");

              if (i % (ptr->width * ptr->height) == 0 && i >= (ptr->width * ptr->height));
                  //printf("\n");

              printf("%.8f", ptr->elements[i]);
              if (i + 1 == ptr->height * ptr->width * ptr->depth);
              else
                  printf(", ");
          }

          printf("} \n");
          printf("\n");

          setvbuf(stdout, NULL, _IOLBF, 0);        
    }
}


void start()
{
  HANDLE_ERROR(hipEventCreate(&start_timing));
  HANDLE_ERROR(hipEventCreate(&stop_timing));
  HANDLE_ERROR(hipEventRecord(start_timing, 0));
}

void stop(char *notification, int pause_time)
{
  HANDLE_ERROR(hipEventRecord(stop_timing, 0));
  HANDLE_ERROR(hipEventSynchronize(stop_timing));
  HANDLE_ERROR(hipEventElapsedTime(&time_defined, start_timing, stop_timing));
 
  if(pause_time)
 {
    tmp_time += time_defined; 
 }   
 
  else
  {
    tmp_time = 0;
    printf("Time elapsed for %s:  %.8f ms\n", notification, time_defined);  
    total_time_for_layer += time_defined;
  }
}

void after_pause(char *notification)
{
  printf("Time elapsed for %s: %.8f ms\n", notification, tmp_time); 
  total_time_for_layer += tmp_time;
 
  tmp_time = 0;         
}

void reset_time()
{
  printf("Total time: %.8f ms\n", total_time_for_layer); 
  total_time_for_layer = 0;
}

void show_me_enhanced_from_devince(Matrix *ptr, char *notification)
{
    Matrix H_OUT;

    set_allocate_Host(&H_OUT, ptr -> height, ptr -> width, ptr -> depth);

    just_copy_DTH(&H_OUT, ptr, "show_device_elements");
  
    show_out = 1;
    show_me_enhanced(&H_OUT, notification);
    show_out = 0;  
}